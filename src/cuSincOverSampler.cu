#include "hip/hip_runtime.h"
/* 
 * cuSincOverSampler.cu
 */
#include "cuArrays.h"
#include "cuSincOverSampler.h"
#include "cuArrays.h"
#include "cudaUtil.h"
#include "hipError_t.h"
#include "cuAmpcorUtil.h"

cuSincOverSamplerR2R::cuSincOverSamplerR2R(const int i_intplength_, const int i_covs_, hipStream_t stream_)
 : i_intplength(i_intplength_), i_covs(i_covs_)
{
    setStream(stream_);
    //i_intplength = int(r_relfiltlen/r_beta);
    r_relfiltlen = r_beta * i_intplength;
    i_filtercoef = i_intplength*i_decfactor;
    r_wgthgt = (1.0f - r_pedestal)/2.0f;
    r_soff = (i_filtercoef)/2.0f;
    r_soff_inverse = 1.0f/r_soff;
    r_decfactor_inverse = 1.0f/i_decfactor;
    checkCudaErrors(hipMalloc((void **)&r_filter, (i_filtercoef+1)*sizeof(float)));
    cuSetupSincKernel();
}

void cuSincOverSamplerR2R::setStream(hipStream_t stream_)
{
    stream = stream_;
}

cuSincOverSamplerR2R::~cuSincOverSamplerR2R() 
{
    checkCudaErrors(hipFree(r_filter));
}


__global__ void cuSetupSincKernel_kernel(float *r_filter_, const int i_filtercoef_, 
    const float r_soff_, const float r_wgthgt_, const int i_weight_,
    const float r_soff_inverse_, const float r_beta_, const float r_decfactor_inverse_,
    const float r_relfiltlen_inverse_)
{
    int i = threadIdx.x + blockDim.x*blockIdx.x;
    if(i > i_filtercoef_) return;
    float r_wa = i - r_soff_;
    float r_wgt = (1.0f - r_wgthgt_) + r_wgthgt_*cos(PI*r_wa*r_soff_inverse_);
    float r_s = r_wa*r_beta_*r_decfactor_inverse_*PI;
    float r_fct; 
    if(r_s != 0.0f) {
        r_fct = sin(r_s)/r_s;
    }
    else {
        r_fct = 1.0f;
    }
    if(i_weight_ == 1) {
        r_filter_[i] = r_fct*r_wgt;
    }
    else {
        r_filter_[i] = r_fct;
    }
    //printf("kernel %d %f\n", i, r_filter_[i]);
}


void cuSincOverSamplerR2R::cuSetupSincKernel()
{
    const int nthreads = 128;
    const int nblocks = IDIVUP(i_filtercoef, nthreads);
    float r_relfiltlen_inverse = 1.0f/r_relfiltlen; 
    cuSetupSincKernel_kernel<<<nblocks, nthreads, 0, stream>>> (
        r_filter, i_filtercoef, r_soff, r_wgthgt, i_weight, 
        r_soff_inverse, r_beta, r_decfactor_inverse, r_relfiltlen_inverse);
    getLastCudaError("cuSetupSincKernel_kernel");
}

__global__ void cuSincInterpolation_kernel(const int nImages, 
    const float * imagesIn, const int inNX, const int inNY,
    float * imagesOut, const int outNX, const int outNY, 
    const float * r_filter_, const int i_covs_, const int i_decfactor_, const int i_intplength_, 
    const int i_startX, const int i_startY, const int i_int_size)
{
    int idxImage = blockIdx.z;
    int idxX = threadIdx.x + blockDim.x*blockIdx.x; 
    int idxY = threadIdx.y + blockDim.y*blockIdx.y;
    if(idxImage >=nImages || idxX >= i_int_size || idxY >= i_int_size) return;
    int outx = idxX + i_startX;
    int outy = idxY + i_startY;
    int idxOut = idxImage*outNX*outNY + outx*outNY + outy;
    
    float r_xout = (float)outx/i_covs_;
    int i_xout = int(r_xout);
    float r_xfrac = r_xout - i_xout;
    int i_xfrac = int(r_xfrac*i_decfactor_);
    
    float r_yout = (float)outy/i_covs_;
    int i_yout = int(r_yout);
    float r_yfrac = r_yout - i_yout;
    int i_yfrac = int(r_yfrac*i_decfactor_);
    
    float intpData = 0.0f;
    float r_sincwgt = 0.0f;
    float r_sinc_coef;
    
    for(int i=0; i < inNX; i++) {
        int i_xindex = i_xout - i + i_intplength_/2;
        if(i_xindex < 0) i_xindex+= i_intplength_;
        if(i_xindex >= i_intplength_) i_xindex-=i_intplength_;  
        float r_xsinc_coef = r_filter_[i_xindex*i_decfactor_+i_xfrac];
        
        for(int j=0; j< inNY; j++) {
            int i_yindex = i_yout - j + i_intplength_/2;
            if(i_yindex < 0) i_yindex+= i_intplength_;
            if(i_yindex >= i_intplength_) i_yindex-=i_intplength_;  
            float r_ysinc_coef = r_filter_[i_yindex*i_decfactor_+i_yfrac];
            r_sinc_coef = r_xsinc_coef*r_ysinc_coef;
            r_sincwgt += r_sinc_coef;
            intpData += imagesIn[idxImage*inNX*inNY+i*inNY+j]*r_sinc_coef;
            /*
              if(outx == 0 && outy == 1) {
                printf("intp kernel %d %d %d %d %d %d %d %f\n", i, j, i_xindex, i_yindex, i_xindex*i_decfactor_+i_xfrac,
                   i_yindex*i_decfactor_+i_yfrac, idxImage*inNX*inNY+i*inNY+j, r_sinc_coef);
              }*/
        }
    }
    imagesOut[idxOut] = intpData/r_sincwgt;
    //printf("test int kernel %d %d %f %f %f\n", outx, outy, intpData, r_sincwgt, imagesOut[idxOut]);
}


void cuSincOverSamplerR2R::execute(cuArrays<float> *imagesIn, cuArrays<float> *imagesOut)
{
    const int nImages = imagesIn->count;
    const int inNX = imagesIn->height;
    const int inNY = imagesIn->width;
    const int outNX = imagesOut->height; 
    const int outNY = imagesOut->width;
    
    const int i_int_range = i_sincwindow * i_covs; 
    const int i_int_startX = outNX/2 - i_int_range;
    const int i_int_startY = outNY/2 - i_int_range;
    const int i_int_size = 2*i_int_range + 1;
      
    imagesOut->setZero(stream);
    
    static const int nthreads = 16;
    dim3 threadsperblock(nthreads, nthreads, 1);
    dim3 blockspergrid (IDIVUP(i_int_size, nthreads), IDIVUP(i_int_size, nthreads), nImages);
    cuSincInterpolation_kernel<<<blockspergrid, threadsperblock, 0, stream>>>(nImages, 
        imagesIn->devData, inNX, inNY,
        imagesOut->devData, outNX, outNY,
        r_filter, i_covs, i_decfactor, i_intplength, i_int_startX, i_int_startY, i_int_size);
    getLastCudaError("cuSincInterpolation_kernel");
}

// end of file



