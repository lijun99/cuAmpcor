#include "hip/hip_runtime.h"
/* 
 * cuOverSampler.cu
 * define cuOverSampler class, to save cufft plans and perform oversampling calculations
 */
#include "cuArrays.h"
#include "cuOverSampler.h"
#include "cuArrays.h"
#include "cudaUtil.h"
#include "hipError_t.h"
#include "cuAmpcorUtil.h"

// Oversampler for complex data 
cuOverSamplerC2C::cuOverSamplerC2C(int inNX, int inNY, int outNX, int outNY, int nImages, hipStream_t stream_)
{
    
    int inNXp2 = inNX;
    int inNYp2 = inNY;
    int outNXp2 = outNX;
    int outNYp2 = outNY;
    
    /* if expanded to 2^n
    int inNXp2 = nextpower2(inNX);
    int inNYp2 = nextpower2(inNY);
    int outNXp2 = inNXp2*outNX/inNX;
    int outNYp2 = inNYp2*outNY/inNY; 
    */
    
    workIn = new cuArrays<float2>(inNXp2, inNYp2, nImages);
    workIn->allocate();
    workOut = new cuArrays<float2>(outNXp2, outNYp2, nImages);
    workOut->allocate();
    int imageSize = inNXp2*inNYp2;
    int n[NRANK] ={inNXp2, inNYp2};
    int fImageSize = inNXp2*inNYp2;
    int nOverSample[NRANK] = {outNXp2, outNYp2};
    int fImageOverSampleSize = outNXp2*outNYp2;
    cufft_Error(hipfftPlanMany(&forwardPlan, NRANK, n, NULL, 1, imageSize, NULL, 1, fImageSize, HIPFFT_C2C, nImages));
    cufft_Error(hipfftPlanMany(&backwardPlan, NRANK, nOverSample, NULL, 1, fImageOverSampleSize, NULL, 1, fImageOverSampleSize, HIPFFT_C2C, nImages));
    setStream(stream_);
}

void cuOverSamplerC2C::setStream(hipStream_t stream_)
{
    this->stream = stream_;
    hipfftSetStream(forwardPlan, stream);
    hipfftSetStream(backwardPlan, stream);
}

//tested
void cuOverSamplerC2C::execute(cuArrays<float2> *imagesIn, cuArrays<float2> *imagesOut)
{
    //cuArraysCopyPadded(imagesIn, workIn, stream);  
    cufft_Error(hipfftExecC2C(forwardPlan, imagesIn->devData, workIn->devData, HIPFFT_BACKWARD));
    cuArraysPaddingMany(workIn, workOut, stream);
    cufft_Error(hipfftExecC2C(backwardPlan, workOut->devData, imagesOut->devData, HIPFFT_FORWARD));
    //cuArraysCopyExtract(workOut, imagesOut, make_int2(0,0), stream);
}

void cuOverSamplerC2C::execute(cuArrays<float2> *imagesIn, cuArrays<float2> *imagesOut, int method)
{   
    cuDeramp(method, imagesIn, stream);         
    cufft_Error(hipfftExecC2C(forwardPlan, imagesIn->devData, workIn->devData, HIPFFT_BACKWARD ));
    cuArraysPaddingMany(workIn, workOut, stream);
    cufft_Error(hipfftExecC2C(backwardPlan, workOut->devData, imagesOut->devData, HIPFFT_FORWARD));
}

cuOverSamplerC2C::~cuOverSamplerC2C() 
{
    cufft_Error(hipfftDestroy(forwardPlan));
    cufft_Error(hipfftDestroy(backwardPlan));
    delete(workIn);
    delete(workOut);	
}


// oversampler for real data
cuOverSamplerR2R::cuOverSamplerR2R(int inNX, int inNY, int outNX, int outNY, int nImages, hipStream_t stream)
{
    
/*    
    int inNXp2 = nextpower2(inNX);
    int inNYp2 = nextpower2(inNY);
    int outNXp2 = inNXp2*outNX/inNX;
    int outNYp2 = inNYp2*outNY/inNY;    
*/
    
    int inNXp2 = inNX;
    int inNYp2 = inNY;
    int outNXp2 = outNX;
    int outNYp2 = outNY;

    int imageSize = inNXp2 *inNYp2;
    int n[NRANK] ={inNXp2, inNYp2};
    int fImageSize = inNXp2*inNYp2;
    int nUpSample[NRANK] = {outNXp2, outNYp2};
    int fImageUpSampleSize = outNXp2*outNYp2;
    workSizeIn = new cuArrays<float2>(inNXp2, inNYp2, nImages);
    workSizeIn->allocate();
    workSizeOut = new cuArrays<float2>(outNXp2, outNYp2, nImages);
    workSizeOut->allocate();
    cufft_Error(hipfftPlanMany(&forwardPlan, NRANK, n, NULL, 1, imageSize, NULL, 1, fImageSize, HIPFFT_C2C, nImages));
    cufft_Error(hipfftPlanMany(&backwardPlan, NRANK, nUpSample, NULL, 1, fImageUpSampleSize, NULL, 1, outNX*outNY, HIPFFT_C2C, nImages));
    setStream(stream);
}

void cuOverSamplerR2R::setStream(hipStream_t stream_)
{
    stream = stream_;
    hipfftSetStream(forwardPlan, stream);
    hipfftSetStream(backwardPlan, stream);
}

//tested
void cuOverSamplerR2R::execute(cuArrays<float> *imagesIn, cuArrays<float> *imagesOut)
{
    cuArraysCopyPadded(imagesIn, workSizeIn, stream);
    cufft_Error(hipfftExecC2C(forwardPlan, workSizeIn->devData, workSizeIn->devData, HIPFFT_BACKWARD));
    cuArraysPaddingMany(workSizeIn, workSizeOut, stream);
    cufft_Error(hipfftExecC2C(backwardPlan, workSizeOut->devData, workSizeOut->devData,HIPFFT_FORWARD ));
    cuArraysCopyExtract(workSizeOut, imagesOut, make_int2(0,0), stream);	
}

cuOverSamplerR2R::~cuOverSamplerR2R() 
{
    cufft_Error(hipfftDestroy(forwardPlan));
    cufft_Error(hipfftDestroy(backwardPlan));	
    workSizeIn->deallocate();
    workSizeOut->deallocate();
}






