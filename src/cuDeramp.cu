#include "hip/hip_runtime.h"
/*
 * @file  cuDeramp.cu
 * @brief Derampling a batch of 2D complex images with GPU
 *
 * A phase ramp is equivalent to a frequency shift in frequency domain,
 *   which needs to be removed (deramping) in order to move the band center
 *   to zero. This is necessary before oversampling a complex signal.
 * Method 1: each signal is decomposed into real and imaginary parts,
 *   and the average phase shift is obtained as atan(\sum imag / \sum real).
 *   The average is weighted by the amplitudes (coherence).
 * Method 0 or else: skip deramping
 *
 */
 
#include "cuArrays.h" 
#include "float2.h" 
#include <cfloat>
#include "hipError_t.h"
#include "cudaUtil.h"
#include "cuAmpcorUtil.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <limits>


// cuda does not have a good support on volatile vector struct, e.g. float2
// have to use regular float type for shared memory (volatile) data
// the following methods are defined to operate float2/complex objects through float
inline static __device__ void copyToShared(volatile float *s, const int i, const float2 x, const int block) 
{ s[i] = x.x; s[i+block] = x.y; }

inline static __device__ void copyFromShared(float2 &x, volatile float *s, const int i, const int block) 
{ x.x = s[i]; x.y = s[i+block]; }


inline static __device__ void addInShared(volatile float *s, const int i, const int j, const int block) 
{ s[i] += s[i+j]; s[i+block] += s[i+j+block];}


// kernel to do sum reduction for float2 within a block
template <const int nthreads>
__device__ void complexSumReduceBlock(float2& sum, volatile float *shmem)
{
    const int tid = threadIdx.x;
    copyToShared(shmem, tid, sum, nthreads);
    __syncthreads();
    
    if (nthreads >=1024) { if (tid < 512) { addInShared(shmem, tid, 512, nthreads); } __syncthreads(); }
    if (nthreads >= 512) { if (tid < 256) { addInShared(shmem, tid, 256, nthreads); } __syncthreads(); }
    if (nthreads >= 256) { if (tid < 128) { addInShared(shmem, tid, 128, nthreads); } __syncthreads(); }
    if (nthreads >= 128) { if (tid <  64) { addInShared(shmem, tid,  64, nthreads); } __syncthreads(); }
    if (tid < 32)
    {	
        addInShared(shmem, tid, 32, nthreads);
        addInShared(shmem, tid, 16, nthreads);
        addInShared(shmem, tid,  8, nthreads);
        addInShared(shmem, tid,  4, nthreads);
        addInShared(shmem, tid,  2, nthreads);
        addInShared(shmem, tid,  1, nthreads); 
    }
    __syncthreads();
    copyFromShared(sum, shmem, 0, nthreads);
}

// cuda kernel for cuDerampMethod1
template<const int nthreads>
__global__ void cuDerampMethod1_kernel(float2 *images, const int imageNX, int const imageNY, 
    const int imageSize, const int nImages, const float normCoef)
{
    __shared__ float shmem[2*nthreads];
    int pixelIdx, pixelIdxX, pixelIdxY;
    
    const int bid = blockIdx.x;    
    if(bid >= nImages) return;
    float2 *image = images+ bid*imageSize;
    const int tid = threadIdx.x;  
    float2 phaseDiffY  = make_float2(0.0f, 0.0f);
    for (int i = tid; i < imageSize; i += nthreads) {
        pixelIdxY = i % imageNY;
        if(pixelIdxY < imageNY -1) {
            pixelIdx = i;
            float2 cprod = complexMulConj( image[pixelIdx], image[pixelIdx+1]);   
            phaseDiffY += cprod;
        } 
    }       
    complexSumReduceBlock<nthreads>(phaseDiffY, shmem);
    //phaseDiffY *= normCoef;
    float phaseY=atan2f(phaseDiffY.y, phaseDiffY.x);

    float2 phaseDiffX  = make_float2(0.0f, 0.0f);
    for (int i = tid; i < imageSize; i += nthreads)  {
        pixelIdxX = i / imageNY; 
        if(pixelIdxX < imageNX -1) {
            pixelIdx = i;
            float2 cprod = complexMulConj(image[i], image[i+imageNY]);
            phaseDiffX += cprod;
        }
    }   
    
    complexSumReduceBlock<nthreads>(phaseDiffX, shmem);
   
    //phaseDiffX *= normCoef;
    float phaseX = atan2f(phaseDiffX.y, phaseDiffX.x);  //+FLT_EPSILON
     
    for (int i = tid; i < imageSize; i += nthreads)
    { 
        pixelIdxX = i%imageNY;
        pixelIdxY = i/imageNY;
        float phase = pixelIdxX*phaseX + pixelIdxY*phaseY;
        float2 phase_factor = make_float2(cosf(phase), sinf(phase));
        image[i] *= phase_factor;
    }     
}

/**
 * Deramp a complex signal with Method 1
 * @brief Each signal is decomposed into real and imaginary parts,
 *   and the average phase shift is obtained as atan(\sum imag / \sum real).
 * @param[inout] images input/output complex signals
 * @param[in] stream cuda stream
 */
void cuDerampMethod1(cuArrays<float2> *images, hipStream_t stream)
{
    
    const dim3 grid(images->count);
    const int imageSize = images->width*images->height;
    const float invSize = 1.0f/imageSize;

    if(imageSize <=64) {
        cuDerampMethod1_kernel<64> <<<grid, 64, 0, stream>>>
        (images->devData, images->height, images->width, 
        imageSize, images->count, invSize); }
     else if(imageSize <=128) {
        cuDerampMethod1_kernel<128> <<<grid, 128, 0, stream>>>
        (images->devData, images->height, images->width, 
        imageSize, images->count, invSize); }   
     else if(imageSize <=256) {
        cuDerampMethod1_kernel<256> <<<grid, 256, 0, stream>>>
        (images->devData, images->height, images->width, 
        imageSize, images->count, invSize); }  
    else  {
        cuDerampMethod1_kernel<512> <<<grid, 512, 0, stream>>>
        (images->devData, images->height, images->width, 
        imageSize, images->count, invSize); }
    getLastCudaError("cuDerampMethod1 kernel error\n");

}
        
void cuDeramp(int method, cuArrays<float2> *images, hipStream_t stream)
{
    switch(method) {
    case 1:
        cuDerampMethod1(images, stream);
        break;
    default:
        break;
    }
}

// end of file