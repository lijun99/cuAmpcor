#include "hip/hip_runtime.h"
#include "cuAmpcorChunk.h"
#include "cuAmpcorUtil.h"

/**
 * Run ampcor process for a batch of images (a chunk)
 * @param[in] idxDown_  index oIDIVUP(i,j) ((i+j-1)/j)f the chunk along Down/Azimuth direction
 * @param[in] idxAcross_ index of the chunk along Across/Range direction
 */
void cuAmpcorChunk::run(int idxDown_, int idxAcross_)
{
    // set chunk index
    setIndex(idxDown_, idxAcross_);

    // load reference image chunk
    loadReferenceChunk();

    //std::cout << "load reference chunk ok\n";

    cuArraysAbs(c_referenceBatchRaw, r_referenceBatchRaw, stream);
    cuArraysSubtractMean(r_referenceBatchRaw, stream);
    // load secondary image chunk
    loadSecondaryChunk();
    cuArraysAbs(c_secondaryBatchRaw, r_secondaryBatchRaw, stream);

    //std::cout << "load secondary chunk ok\n";


    //cross correlation for none-oversampled data
    if(param->algorithm == 0) {
        cuCorrFreqDomain->execute(r_referenceBatchRaw, r_secondaryBatchRaw, r_corrBatchRaw);
    }
    else {
        cuCorrTimeDomain(r_referenceBatchRaw, r_secondaryBatchRaw, r_corrBatchRaw, stream); //time domain cross correlation
    }
    cuCorrNormalize(r_referenceBatchRaw, r_secondaryBatchRaw, r_corrBatchRaw, stream);


    // find the maximum location of none-oversampled correlation
    // 41 x 41, if halfsearchrange=20
    //cuArraysMaxloc2D(r_corrBatchRaw, offsetInit, stream);
    cuArraysMaxloc2D(r_corrBatchRaw, offsetInit, r_maxval, stream);

    offsetInit->outputToFile("offsetInit1", stream);

    // Estimation of statistics
    // Author: Minyan Zhong
    // Extraction of correlation surface around the peak
    cuArraysCopyExtractCorr(r_corrBatchRaw, r_corrBatchRawZoomIn, i_corrBatchZoomInValid, offsetInit, stream);

    hipDeviceSynchronize();

    // debug: output the intermediate results
    r_maxval->outputToFile("r_maxval",stream);
    r_corrBatchRaw->outputToFile("r_corrBatchRaw",stream);
    r_corrBatchRawZoomIn->outputToFile("r_corrBatchRawZoomIn",stream);
    i_corrBatchZoomInValid->outputToFile("i_corrBatchZoomInValid",stream);

    // Summation of correlation and data point values
    cuArraysSumCorr(r_corrBatchRawZoomIn, i_corrBatchZoomInValid, r_corrBatchSum, i_corrBatchValidCount, stream);

    // SNR
    cuEstimateSnr(r_corrBatchSum, i_corrBatchValidCount, r_maxval, r_snrValue, stream);

    // Variance
    // cuEstimateVariance(r_corrBatchRaw, offsetInit, r_maxval, r_covValue, stream);

    // Using the approximate estimation to adjust secondary image (half search window size becomes only 4 pixels)
    //offsetInit->debuginfo(stream);
    // determine the starting pixel to extract secondary images around the max location
    cuDetermineSecondaryExtractOffset(offsetInit,
        param->halfSearchRangeDownRaw, // old range
        param->halfSearchRangeAcrossRaw,
        param->halfZoomWindowSizeRaw,  // new range
        param->halfZoomWindowSizeRaw,
        stream);
    //offsetInit->debuginfo(stream);
    // oversample reference
    // (deramping now included in oversampler)
    referenceBatchOverSampler->execute(c_referenceBatchRaw, c_referenceBatchOverSampled, param->derampMethod);
    cuArraysAbs(c_referenceBatchOverSampled, r_referenceBatchOverSampled, stream);
    cuArraysSubtractMean(r_referenceBatchOverSampled, stream);

    // extract secondary and oversample
    cuArraysCopyExtract(c_secondaryBatchRaw, c_secondaryBatchZoomIn, offsetInit, stream);
    secondaryBatchOverSampler->execute(c_secondaryBatchZoomIn, c_secondaryBatchOverSampled, param->derampMethod);
    cuArraysAbs(c_secondaryBatchOverSampled, r_secondaryBatchOverSampled, stream);

    // correlate oversampled images
    if(param->algorithm == 0) {
        cuCorrFreqDomain_OverSampled->execute(r_referenceBatchOverSampled, r_secondaryBatchOverSampled, r_corrBatchZoomIn);
    }
    else {
        cuCorrTimeDomain(r_referenceBatchOverSampled, r_secondaryBatchOverSampled, r_corrBatchZoomIn, stream);
    }
    cuCorrNormalize(r_referenceBatchOverSampled, r_secondaryBatchOverSampled, r_corrBatchZoomIn, stream);

    //std::cout << "debug correlation oversample\n";
    //std::cout << r_referenceBatchOverSampled->height << " " << r_referenceBatchOverSampled->width << "\n";
    //std::cout << r_secondaryBatchOverSampled->height << " " << r_secondaryBatchOverSampled->width << "\n";
    //std::cout << r_corrBatchZoomIn->height << " " << r_corrBatchZoomIn->width << "\n";

    // oversample the correlation surface
    cuArraysCopyExtract(r_corrBatchZoomIn, r_corrBatchZoomInAdjust, make_int2(0,0), stream);

    //std::cout << "debug oversampling " << r_corrBatchZoomInAdjust << " " << r_corrBatchZoomInOverSampled << "\n";

    if(param->oversamplingMethod) {
        corrSincOverSampler->execute(r_corrBatchZoomInAdjust, r_corrBatchZoomInOverSampled);
    }
    else {
        corrOverSampler->execute(r_corrBatchZoomInAdjust, r_corrBatchZoomInOverSampled);
    }

    //find the max again

    cuArraysMaxloc2D(r_corrBatchZoomInOverSampled, offsetZoomIn, corrMaxValue, stream);

    // determine the final offset from non-oversampled (pixel) and oversampled (sub-pixel)
    cuSubPixelOffset(offsetInit, offsetZoomIn, offsetFinal,
        param->oversamplingFactor, param->rawDataOversamplingFactor,
        param->halfSearchRangeDownRaw, param->halfSearchRangeAcrossRaw,
        param->halfZoomWindowSizeRaw, param->halfZoomWindowSizeRaw,
        stream);
    //offsetInit->debuginfo(stream);
    //offsetZoomIn->debuginfo(stream);
    //offsetFinal->debuginfo(stream);

    // Do insertion.
    // Offsetfields.
    cuArraysCopyInsert(offsetFinal, offsetImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);

    // Debugging matrix.
    cuArraysCopyInsert(r_corrBatchSum, floatImage1, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
    cuArraysCopyInsert(i_corrBatchValidCount, intImage1, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);

    // Old: save max correlation coefficients.
    //cuArraysCopyInsert(corrMaxValue, snrImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
    // New: save SNR
    cuArraysCopyInsert(r_snrValue, snrImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);

    // Variance.
    cuArraysCopyInsert(r_covValue, covImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
}

void cuAmpcorChunk::setIndex(int idxDown_, int idxAcross_)
{
    idxChunkDown = idxDown_;
    idxChunkAcross = idxAcross_;
	idxChunk = idxChunkAcross + idxChunkDown*param->numberChunkAcross;

    if(idxChunkDown == param->numberChunkDown -1) {
		nWindowsDown = param->numberWindowDown - param->numberWindowDownInChunk*(param->numberChunkDown -1);
	}
	else {
		nWindowsDown = param->numberWindowDownInChunk;
	}

	if(idxChunkAcross == param->numberChunkAcross -1) {
		nWindowsAcross = param->numberWindowAcross - param->numberWindowAcrossInChunk*(param->numberChunkAcross -1);
	}
	else {
		nWindowsAcross = param->numberWindowAcrossInChunk;
	}
	//std::cout << "DEBUG setIndex" << idxChunk << " " << nWindowsDown << " " << nWindowsAcross << "\n";

}

/// obtain the starting pixels for each chip
/// @param[in] oStartPixel
///
void cuAmpcorChunk::getRelativeOffset(int *rStartPixel, const int *oStartPixel, int diff)
{
    for(int i=0; i<param->numberWindowDownInChunk; ++i) {
		int iDown = i;
		if(i>=nWindowsDown) iDown = nWindowsDown-1;
        for(int j=0; j<param->numberWindowAcrossInChunk; ++j){
			int iAcross = j;
			if(j>=nWindowsAcross) iAcross = nWindowsAcross-1;
            int idxInChunk = iDown*param->numberWindowAcrossInChunk+iAcross;
            int idxInAll = (iDown+idxChunkDown*param->numberWindowDownInChunk)*param->numberWindowAcross
				+ idxChunkAcross*param->numberWindowAcrossInChunk+iAcross;
            rStartPixel[idxInChunk] = oStartPixel[idxInAll] - diff;
            //fprintf(stderr, "relative offset %d %d %d %d\n", i, j, rStartPixel[idxInChunk], diff);
        }
    }
}

void cuAmpcorChunk::loadReferenceChunk()
{

    // we first load the whole chunk of image from cpu to a gpu buffer c(r)_referenceChunkRaw
    // then copy to a batch of windows with (nImages, height, width) (leading dimension on the right)

    // get the chunk size to be loaded to gpu
    int startD = param->referenceChunkStartPixelDown[idxChunk]; //start pixel down (along height)
    int startA = param->referenceChunkStartPixelAcross[idxChunk]; // start pixel across (along width)
    int height =  param->referenceChunkHeight[idxChunk]; // number of pixels along height
    int width = param->referenceChunkWidth[idxChunk];  // number of pixels along width

    //use cpu to compute the starting positions for each window
    getRelativeOffset(ChunkOffsetDown->hostData, param->referenceStartPixelDown, param->referenceChunkStartPixelDown[idxChunk]);
    // copy the positions to gpu
    ChunkOffsetDown->copyToDevice(stream);
    // same for the across direction
    getRelativeOffset(ChunkOffsetAcross->hostData, param->referenceStartPixelAcross, param->referenceChunkStartPixelAcross[idxChunk]);
    ChunkOffsetAcross->copyToDevice(stream);

    // check whether the image is complex (e.g., SLC) or real( e.g. TIFF)
    if(referenceImage->isComplex())
    {
        // allocate a gpu buffer to load data from cpu/file
        // try allocate/deallocate the buffer on the fly to save gpu memory 07/09/19
        c_referenceChunkRaw = new cuArrays<float2> (param->maxReferenceChunkHeight, param->maxReferenceChunkWidth);
        c_referenceChunkRaw->allocate();

        // load the data from cpu
        referenceImage->loadToDevice((void *)c_referenceChunkRaw->devData, startD, startA, height, width, stream);
        //std::cout << "debug load reference: " << startD << " " <<  startA << " " <<  height << " "  << width << "\n";

        //copy the chunk to a batch format (nImages, height, width)
        // if derampMethod = 0 (no deramp), take amplitudes; otherwise, copy complex data
        if(param->derampMethod == 0) {
            cuArraysCopyToBatchAbsWithOffset(c_referenceChunkRaw, param->referenceChunkWidth[idxChunk],
                c_referenceBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        }
        else {
            cuArraysCopyToBatchWithOffset(c_referenceChunkRaw, param->referenceChunkWidth[idxChunk],
                c_referenceBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        }
        // deallocate the gpu buffer
        delete c_referenceChunkRaw;
    }
    // if the image is real
    else {
        r_referenceChunkRaw = new cuArrays<float> (param->maxReferenceChunkHeight, param->maxReferenceChunkWidth);
        r_referenceChunkRaw->allocate();

        // load the data from cpu
        referenceImage->loadToDevice((void *)r_referenceChunkRaw->devData, startD, startA, height, width, stream);

        // copy the chunk (real) to a batch format (complex)
        cuArraysCopyToBatchWithOffsetR2C(r_referenceChunkRaw, param->referenceChunkWidth[idxChunk],
                c_referenceBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        // deallocate the gpu buffer
        delete r_referenceChunkRaw;
    }


}

void cuAmpcorChunk::loadSecondaryChunk()
{

    //copy to a batch format (nImages, height, width)
    getRelativeOffset(ChunkOffsetDown->hostData, param->secondaryStartPixelDown, param->secondaryChunkStartPixelDown[idxChunk]);
    ChunkOffsetDown->copyToDevice(stream);
    getRelativeOffset(ChunkOffsetAcross->hostData, param->secondaryStartPixelAcross, param->secondaryChunkStartPixelAcross[idxChunk]);
    ChunkOffsetAcross->copyToDevice(stream);

    if(secondaryImage->isComplex())
    {
        c_secondaryChunkRaw = new cuArrays<float2> (param->maxSecondaryChunkHeight, param->maxSecondaryChunkWidth);
        c_secondaryChunkRaw->allocate();

        //load a chunk from mmap to gpu
        secondaryImage->loadToDevice(c_secondaryChunkRaw->devData,
            param->secondaryChunkStartPixelDown[idxChunk],
            param->secondaryChunkStartPixelAcross[idxChunk],
            param->secondaryChunkHeight[idxChunk],
            param->secondaryChunkWidth[idxChunk],
            stream);

        if(param->derampMethod == 0) {
            cuArraysCopyToBatchAbsWithOffset(c_secondaryChunkRaw, param->secondaryChunkWidth[idxChunk],
                c_secondaryBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        }
        else {
           cuArraysCopyToBatchWithOffset(c_secondaryChunkRaw, param->secondaryChunkWidth[idxChunk],
                c_secondaryBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        }
        delete c_secondaryChunkRaw;
    }
    else { //real image
        //allocate the gpu buffer
        r_secondaryChunkRaw = new cuArrays<float> (param->maxSecondaryChunkHeight, param->maxSecondaryChunkWidth);
        r_secondaryChunkRaw->allocate();

        //load a chunk from mmap to gpu
        secondaryImage->loadToDevice(r_secondaryChunkRaw->devData,
            param->secondaryChunkStartPixelDown[idxChunk],
            param->secondaryChunkStartPixelAcross[idxChunk],
            param->secondaryChunkHeight[idxChunk],
            param->secondaryChunkWidth[idxChunk],
            stream);

        // convert to the batch format
        cuArraysCopyToBatchWithOffsetR2C(r_secondaryChunkRaw, param->secondaryChunkWidth[idxChunk],
                c_secondaryBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        delete r_secondaryChunkRaw;
    }
}

cuAmpcorChunk::cuAmpcorChunk(cuAmpcorParameter *param_, GDALImage *reference_, GDALImage *secondary_,
    cuArrays<float2> *offsetImage_, cuArrays<float> *snrImage_, cuArrays<float3> *covImage_, cuArrays<int> *intImage1_, cuArrays<float> *floatImage1_, hipStream_t stream_)

{
    param = param_;
    referenceImage = reference_;
    secondaryImage = secondary_;
    offsetImage = offsetImage_;
    snrImage = snrImage_;
    covImage = covImage_;

    intImage1 = intImage1_;
    floatImage1 = floatImage1_;

    stream = stream_;

    // std::cout << "debug Chunk creator " << param->maxReferenceChunkHeight << " " << param->maxReferenceChunkWidth << "\n";
    // try allocate/deallocate on the fly to save gpu memory 07/09/19
    // c_referenceChunkRaw = new cuArrays<float2> (param->maxReferenceChunkHeight, param->maxReferenceChunkWidth);
    // c_referenceChunkRaw->allocate();

    // c_secondaryChunkRaw = new cuArrays<float2> (param->maxSecondaryChunkHeight, param->maxSecondaryChunkWidth);
    // c_secondaryChunkRaw->allocate();

    ChunkOffsetDown = new cuArrays<int> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    ChunkOffsetDown->allocate();
    ChunkOffsetDown->allocateHost();
    ChunkOffsetAcross = new cuArrays<int> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    ChunkOffsetAcross->allocate();
    ChunkOffsetAcross->allocateHost();

    c_referenceBatchRaw = new cuArrays<float2> (
        param->windowSizeHeightRaw, param->windowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_referenceBatchRaw->allocate();

    c_secondaryBatchRaw = new cuArrays<float2> (
        param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_secondaryBatchRaw->allocate();

    r_referenceBatchRaw = new cuArrays<float> (
        param->windowSizeHeightRaw, param->windowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_referenceBatchRaw->allocate();

    r_secondaryBatchRaw = new cuArrays<float> (
        param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_secondaryBatchRaw->allocate();

    c_secondaryBatchZoomIn = new cuArrays<float2> (
        param->searchWindowSizeHeightRawZoomIn, param->searchWindowSizeWidthRawZoomIn,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_secondaryBatchZoomIn->allocate();

    c_referenceBatchOverSampled = new cuArrays<float2> (
			param->windowSizeHeight, param->windowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_referenceBatchOverSampled->allocate();

    c_secondaryBatchOverSampled = new cuArrays<float2> (
			param->searchWindowSizeHeight, param->searchWindowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_secondaryBatchOverSampled->allocate();

    r_referenceBatchOverSampled = new cuArrays<float> (
			param->windowSizeHeight, param->windowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_referenceBatchOverSampled->allocate();

    r_secondaryBatchOverSampled = new cuArrays<float> (
			param->searchWindowSizeHeight, param->searchWindowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_secondaryBatchOverSampled->allocate();

    referenceBatchOverSampler = new cuOverSamplerC2C(
        c_referenceBatchRaw->height, c_referenceBatchRaw->width, //orignal size
        c_referenceBatchOverSampled->height, c_referenceBatchOverSampled->width, //oversampled size
        c_referenceBatchRaw->count, stream);

    secondaryBatchOverSampler = new cuOverSamplerC2C(c_secondaryBatchZoomIn->height, c_secondaryBatchZoomIn->width,
            c_secondaryBatchOverSampled->height, c_secondaryBatchOverSampled->width, c_secondaryBatchRaw->count, stream);

    r_corrBatchRaw = new cuArrays<float> (
			param->searchWindowSizeHeightRaw-param->windowSizeHeightRaw+1,
			param->searchWindowSizeWidthRaw-param->windowSizeWidthRaw+1,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    r_corrBatchRaw->allocate();

    r_corrBatchZoomIn = new cuArrays<float> (
			param->searchWindowSizeHeight - param->windowSizeHeight+1,
			param->searchWindowSizeWidth - param->windowSizeWidth+1,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    r_corrBatchZoomIn->allocate();

    r_corrBatchZoomInAdjust = new cuArrays<float> (
			param->searchWindowSizeHeight - param->windowSizeHeight,
			param->searchWindowSizeWidth - param->windowSizeWidth,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    r_corrBatchZoomInAdjust->allocate();


    r_corrBatchZoomInOverSampled = new cuArrays<float> (
        param->zoomWindowSize * param->oversamplingFactor,
        param->zoomWindowSize * param->oversamplingFactor,
        param->numberWindowDownInChunk,
        param->numberWindowAcrossInChunk);
    r_corrBatchZoomInOverSampled->allocate();

    offsetInit = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetInit->allocate();

    offsetZoomIn = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetZoomIn->allocate();

    offsetFinal = new cuArrays<float2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetFinal->allocate();

    corrMaxValue = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    corrMaxValue->allocate();


    // new arrays due to snr estimation
    std::cout<< "corrRawZoomInHeight: " << param->corrRawZoomInHeight << "\n";
    std::cout<< "corrRawZoomInWidth: " << param->corrRawZoomInWidth << "\n";

    r_corrBatchRawZoomIn = new cuArrays<float> (
			param->corrRawZoomInHeight,
			param->corrRawZoomInWidth,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    r_corrBatchRawZoomIn->allocate();

    i_corrBatchZoomInValid = new cuArrays<int> (
			param->corrRawZoomInHeight,
			param->corrRawZoomInWidth,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    i_corrBatchZoomInValid->allocate();


    r_corrBatchSum = new cuArrays<float> (
                    param->numberWindowDownInChunk,
                    param->numberWindowAcrossInChunk);
    r_corrBatchSum->allocate();

    i_corrBatchValidCount = new cuArrays<int> (
                        param->numberWindowDownInChunk,
                        param->numberWindowAcrossInChunk);
    i_corrBatchValidCount->allocate();

    i_maxloc = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    i_maxloc->allocate();

    r_maxval = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_maxval->allocate();

    r_snrValue = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_snrValue->allocate();

    r_covValue = new cuArrays<float3> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_covValue->allocate();

    // end of new arrays

    if(param->oversamplingMethod) {
        corrSincOverSampler = new cuSincOverSamplerR2R(param->zoomWindowSize, param->oversamplingFactor, stream);
    }
    else {
        corrOverSampler= new cuOverSamplerR2R(param->zoomWindowSize, param->zoomWindowSize,
			(param->zoomWindowSize)*param->oversamplingFactor,
		    (param->zoomWindowSize)*param->oversamplingFactor,
		    param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
		    stream);
	}
    if(param->algorithm == 0) {
        cuCorrFreqDomain = new cuFreqCorrelator(
            param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
            param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
            stream);
        cuCorrFreqDomain_OverSampled = new cuFreqCorrelator(
            param->searchWindowSizeHeight, param->searchWindowSizeWidth,
            param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
            stream);
    }



    debugmsg("all objects in chunk are created ...\n");

}
cuAmpcorChunk::~cuAmpcorChunk()
{
    /*
    delete referenceChunkRaw;
    delete secondaryChunkRaw;
    delete ChunkOffsetDown;
    delete ChunkOffsetAcross;
    delete referenceBatchRaw;
    delete secondaryBatchRaw;
    delete referenceChunkOverSampled;
    delete secondaryChunkOverSampled;
    delete referenceChunkOverSampler;
    delete secondaryChunkOverSampler;
    delete referenceChunk;
    delete secondaryChunk;
    delete corrChunk;
    delete offsetInit;
    delete zoomInOffset;
    delete offsetFinal;
    delete corrChunkZoomIn;
    delete corrChunkZoomInOverSampled;
    delete corrOverSampler;
    delete corrSincOverSampler;
    delete corrMaxValue;
    if(param->algorithm == 0)
        delete cuCorrFreqDomain;
    */
}
