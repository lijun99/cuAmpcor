#include "hip/hip_runtime.h"
#include "cuAmpcorChunk.h"
#include "cuAmpcorUtil.h"

/**
 * Run ampcor process for a batch of images (a chunk) 
 * @param[in] idxDown_  index oIDIVUP(i,j) ((i+j-1)/j)f the chunk along Down/Azimuth direction
 * @param[in] idxAcross_ index of the chunk along Across/Range direction
 */ 
void cuAmpcorChunk::run(int idxDown_, int idxAcross_)
{
    // set chunk index
    setIndex(idxDown_, idxAcross_);
    
    // load master image chunk 
    loadMasterChunk(); 
    
    //std::cout << "load master chunk ok\n";
    
    cuArraysAbs(c_masterBatchRaw, r_masterBatchRaw, stream);
    cuArraysSubtractMean(r_masterBatchRaw, stream);
    // load slave image chunk
    loadSlaveChunk();
    cuArraysAbs(c_slaveBatchRaw, r_slaveBatchRaw, stream);
    
    //std::cout << "load slave chunk ok\n";
    
    
    //cross correlation for none-oversampled data
    if(param->algorithm == 0) {
        cuCorrFreqDomain->execute(r_masterBatchRaw, r_slaveBatchRaw, r_corrBatchRaw);
    }
    else {
        cuCorrTimeDomain(r_masterBatchRaw, r_slaveBatchRaw, r_corrBatchRaw, stream); //time domain cross correlation
    }    
    cuCorrNormalize(r_masterBatchRaw, r_slaveBatchRaw, r_corrBatchRaw, stream);
    //find the maximum location of none-oversampled correlation
    cuArraysMaxloc2D(r_corrBatchRaw, offsetInit, stream);

// Estimate SNR (Minyan Zhong)

    //std::cout<< "flag stats 1" <<std::endl; 
    //cuArraysCopyExtractCorr(r_corrBatchRaw, r_corrBatchZoomIn, i_corrBatchZoomInValid, offsetInit, stream);

    //std::cout<< "flag stats 2" <<std::endl;
    //cuArraysSumCorr(r_corrBatchZoomIn, i_corrBatchZoomInValid, r_corrBatchSum, i_corrBatchValidCount, stream);

    //std::cout<< "flag stats 3" <<std::endl;
    //cuEstimateSnr(r_corrBatchSum, i_corrBatchValidCount, r_maxval, r_snrValue, stream);

//
    
    //offsetInit->debuginfo(stream);
    // determine the starting pixel to extract slave images around the max location
    cuDetermineSlaveExtractOffset(offsetInit, 
        param->halfSearchRangeDownRaw, // old range
        param->halfSearchRangeAcrossRaw, 
        param->halfZoomWindowSizeRaw,  // new range
        param->halfZoomWindowSizeRaw,
        stream);
    //offsetInit->debuginfo(stream);
    // oversample master
    // (deramping now included in oversampler)
    masterBatchOverSampler->execute(c_masterBatchRaw, c_masterBatchOverSampled, param->derampMethod);
    cuArraysAbs(c_masterBatchOverSampled, r_masterBatchOverSampled, stream);
    cuArraysSubtractMean(r_masterBatchOverSampled, stream);
    
    // extract slave and oversample
    cuArraysCopyExtract(c_slaveBatchRaw, c_slaveBatchZoomIn, offsetInit, stream);
    slaveBatchOverSampler->execute(c_slaveBatchZoomIn, c_slaveBatchOverSampled, param->derampMethod);
    cuArraysAbs(c_slaveBatchOverSampled, r_slaveBatchOverSampled, stream);
    
    // correlate oversampled images
    if(param->algorithm == 0) {
        cuCorrFreqDomain_OverSampled->execute(r_masterBatchOverSampled, r_slaveBatchOverSampled, r_corrBatchZoomIn);
    }
    else {
        cuCorrTimeDomain(r_masterBatchOverSampled, r_slaveBatchOverSampled, r_corrBatchZoomIn, stream); 
    }      
    cuCorrNormalize(r_masterBatchOverSampled, r_slaveBatchOverSampled, r_corrBatchZoomIn, stream);
    
    //std::cout << "debug correlation oversample\n";
    //std::cout << r_masterBatchOverSampled->height << " " << r_masterBatchOverSampled->width << "\n";
    //std::cout << r_slaveBatchOverSampled->height << " " << r_slaveBatchOverSampled->width << "\n";
    //std::cout << r_corrBatchZoomIn->height << " " << r_corrBatchZoomIn->width << "\n";
    
    // oversample the correlation surface    
    cuArraysCopyExtract(r_corrBatchZoomIn, r_corrBatchZoomInAdjust, make_int2(0,0), stream);
    
    //std::cout << "debug oversampling " << r_corrBatchZoomInAdjust << " " << r_corrBatchZoomInOverSampled << "\n";
    
    if(param->oversamplingMethod) {
        corrSincOverSampler->execute(r_corrBatchZoomInAdjust, r_corrBatchZoomInOverSampled);
    }
    else {
        corrOverSampler->execute(r_corrBatchZoomInAdjust, r_corrBatchZoomInOverSampled); 
    }
    
    //find the max again
    
    cuArraysMaxloc2D(r_corrBatchZoomInOverSampled, offsetZoomIn, corrMaxValue, stream);
    
    // determine the final offset from non-oversampled (pixel) and oversampled (sub-pixel)    
    cuSubPixelOffset(offsetInit, offsetZoomIn, offsetFinal, 
        param->oversamplingFactor, param->rawDataOversamplingFactor, 
        param->halfSearchRangeDownRaw, param->halfSearchRangeAcrossRaw,
        param->halfZoomWindowSizeRaw, param->halfZoomWindowSizeRaw,
        stream);
    //offsetInit->debuginfo(stream);
    //offsetZoomIn->debuginfo(stream);
    //offsetFinal->debuginfo(stream);    
        
    cuArraysCopyInsert(offsetFinal, offsetImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);

    // Minyan Zhong
    //cuArraysCopyInsert(corrMaxValue, snrImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);    
    //cuArraysCopyInsert(r_snrValue, snrImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);    

}

void cuAmpcorChunk::setIndex(int idxDown_, int idxAcross_)
{
    idxChunkDown = idxDown_;
    idxChunkAcross = idxAcross_;
	idxChunk = idxChunkAcross + idxChunkDown*param->numberChunkAcross;
    
    if(idxChunkDown == param->numberChunkDown -1) {
		nWindowsDown = param->numberWindowDown - param->numberWindowDownInChunk*(param->numberChunkDown -1);
	}
	else {
		nWindowsDown = param->numberWindowDownInChunk;
	}
	
	if(idxChunkAcross == param->numberChunkAcross -1) {
		nWindowsAcross = param->numberWindowAcross - param->numberWindowAcrossInChunk*(param->numberChunkAcross -1);
	}
	else {
		nWindowsAcross = param->numberWindowAcrossInChunk;
	}
	//std::cout << "DEBUG setIndex" << idxChunk << " " << nWindowsDown << " " << nWindowsAcross << "\n";
	
}

/// obtain the starting pixels for each chip
/// @param[in] oStartPixel 
///
void cuAmpcorChunk::getRelativeOffset(int *rStartPixel, const int *oStartPixel, int diff)
{
    for(int i=0; i<param->numberWindowDownInChunk; ++i) {
		int iDown = i;
		if(i>=nWindowsDown) iDown = nWindowsDown-1; 
        for(int j=0; j<param->numberWindowAcrossInChunk; ++j){
			int iAcross = j;
			if(j>=nWindowsAcross) iAcross = nWindowsAcross-1; 
            int idxInChunk = iDown*param->numberWindowAcrossInChunk+iAcross;
            int idxInAll = (iDown+idxChunkDown*param->numberWindowDownInChunk)*param->numberWindowAcross
				+ idxChunkAcross*param->numberWindowAcrossInChunk+iAcross;
            rStartPixel[idxInChunk] = oStartPixel[idxInAll] - diff;
            //fprintf(stderr, "relative offset %d %d %d %d\n", i, j, rStartPixel[idxInChunk], diff);
        }
    }
} 

void cuAmpcorChunk::loadMasterChunk()
{
    //load a chunk from mmap to gpu
    int startD = param->masterChunkStartPixelDown[idxChunk];
    int startA = param->masterChunkStartPixelAcross[idxChunk];
    int height =  param->masterChunkHeight[idxChunk];
    int width = param->masterChunkWidth[idxChunk];
    masterImage->loadToDevice(c_masterChunkRaw->devData, startD, startA, height, width, stream);
    std::cout << "debug load master: " << startD << " " <<  startA << " " <<  height << " "  << width << "\n";
    //copy the chunk to a batch of images format (nImages, height, width) 
    //use cpu for some simple math  
    getRelativeOffset(ChunkOffsetDown->hostData, param->masterStartPixelDown, param->masterChunkStartPixelDown[idxChunk]);
    ChunkOffsetDown->copyToDevice(stream);
    getRelativeOffset(ChunkOffsetAcross->hostData, param->masterStartPixelAcross, param->masterChunkStartPixelAcross[idxChunk]);
    ChunkOffsetAcross->copyToDevice(stream);
    // if derampMethod = 0 (no deramp), take amplitudes; otherwise, copy complex data 
    if(param->derampMethod == 0) {
        cuArraysCopyToBatchAbsWithOffset(c_masterChunkRaw, param->masterChunkWidth[idxChunk], 
            c_masterBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
    }
    else {
        cuArraysCopyToBatchWithOffset(c_masterChunkRaw, param->masterChunkWidth[idxChunk], 
            c_masterBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
    }
}

void cuAmpcorChunk::loadSlaveChunk()
{
         //load a chunk from mmap to gpu
    slaveImage->loadToDevice(c_slaveChunkRaw->devData, 
        param->slaveChunkStartPixelDown[idxChunk], 
        param->slaveChunkStartPixelAcross[idxChunk], 
        param->slaveChunkHeight[idxChunk], 
        param->slaveChunkWidth[idxChunk], 
        stream);
    //copy to a batch format (nImages, height, width)
    getRelativeOffset(ChunkOffsetDown->hostData, param->slaveStartPixelDown, param->slaveChunkStartPixelDown[idxChunk]);
    ChunkOffsetDown->copyToDevice(stream);
    getRelativeOffset(ChunkOffsetAcross->hostData, param->slaveStartPixelAcross, param->slaveChunkStartPixelAcross[idxChunk]);
    ChunkOffsetAcross->copyToDevice(stream);
    if(param->derampMethod == 0) {
    cuArraysCopyToBatchAbsWithOffset(c_slaveChunkRaw, param->slaveChunkWidth[idxChunk], 
        c_slaveBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
    }
    else
    {
       cuArraysCopyToBatchWithOffset(c_slaveChunkRaw, param->slaveChunkWidth[idxChunk], 
        c_slaveBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
    } 
}

cuAmpcorChunk::cuAmpcorChunk(cuAmpcorParameter *param_, SlcImage *master_, SlcImage *slave_, 
    cuArrays<float2> *offsetImage_, cuArrays<float> *snrImage_, hipStream_t stream_)
{
    param = param_;
    masterImage = master_;
    slaveImage = slave_;	
    offsetImage = offsetImage_;
    snrImage = snrImage_;
    stream = stream_;
   
    std::cout << "debug Chunk creator " << param->maxMasterChunkHeight << " " << param->maxMasterChunkWidth << "\n";
    c_masterChunkRaw = new cuArrays<float2> (param->maxMasterChunkHeight, param->maxMasterChunkWidth); 
    c_masterChunkRaw->allocate();
    
    c_slaveChunkRaw = new cuArrays<float2> (param->maxSlaveChunkHeight, param->maxSlaveChunkWidth); 
    c_slaveChunkRaw->allocate();
    
    ChunkOffsetDown = new cuArrays<int> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    ChunkOffsetDown->allocate();
    ChunkOffsetDown->allocateHost();
    ChunkOffsetAcross = new cuArrays<int> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    ChunkOffsetAcross->allocate();
    ChunkOffsetAcross->allocateHost();
	
    c_masterBatchRaw = new cuArrays<float2> (
        param->windowSizeHeightRaw, param->windowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_masterBatchRaw->allocate();
    
    c_slaveBatchRaw = new cuArrays<float2> (
        param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_slaveBatchRaw->allocate();
    
    r_masterBatchRaw = new cuArrays<float> (
        param->windowSizeHeightRaw, param->windowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_masterBatchRaw->allocate();
    
    r_slaveBatchRaw = new cuArrays<float> (
        param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_slaveBatchRaw->allocate();
    
    c_slaveBatchZoomIn = new cuArrays<float2> (
        param->searchWindowSizeHeightRawZoomIn, param->searchWindowSizeWidthRawZoomIn,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_slaveBatchZoomIn->allocate();
	
    c_masterBatchOverSampled = new cuArrays<float2> (
			param->windowSizeHeight, param->windowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_masterBatchOverSampled->allocate();

    c_slaveBatchOverSampled = new cuArrays<float2> (
			param->searchWindowSizeHeight, param->searchWindowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_slaveBatchOverSampled->allocate();
    
    r_masterBatchOverSampled = new cuArrays<float> (
			param->windowSizeHeight, param->windowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_masterBatchOverSampled->allocate();

    r_slaveBatchOverSampled = new cuArrays<float> (
			param->searchWindowSizeHeight, param->searchWindowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_slaveBatchOverSampled->allocate();
    
    masterBatchOverSampler = new cuOverSamplerC2C(
        c_masterBatchRaw->height, c_masterBatchRaw->width, //orignal size
        c_masterBatchOverSampled->height, c_masterBatchOverSampled->width, //oversampled size 
        c_masterBatchRaw->count, stream);
	
    slaveBatchOverSampler = new cuOverSamplerC2C(c_slaveBatchZoomIn->height, c_slaveBatchZoomIn->width, 
            c_slaveBatchOverSampled->height, c_slaveBatchOverSampled->width, c_slaveBatchRaw->count, stream);
    
    r_corrBatchRaw = new cuArrays<float> (
			param->searchWindowSizeHeightRaw-param->windowSizeHeightRaw+1, 
			param->searchWindowSizeWidthRaw-param->windowSizeWidthRaw+1, 
			param->numberWindowDownInChunk, 
			param->numberWindowAcrossInChunk);
    r_corrBatchRaw->allocate();
    
    r_corrBatchZoomIn = new cuArrays<float> (
			param->searchWindowSizeHeight - param->windowSizeHeight+1, 
			param->searchWindowSizeWidth - param->windowSizeWidth+1, 
			param->numberWindowDownInChunk, 
			param->numberWindowAcrossInChunk);
    r_corrBatchZoomIn->allocate();
    
    r_corrBatchZoomInAdjust = new cuArrays<float> (
			param->searchWindowSizeHeight - param->windowSizeHeight, 
			param->searchWindowSizeWidth - param->windowSizeWidth, 
			param->numberWindowDownInChunk, 
			param->numberWindowAcrossInChunk);
    r_corrBatchZoomInAdjust->allocate();
    
    
    r_corrBatchZoomInOverSampled = new cuArrays<float> (
        param->zoomWindowSize * param->oversamplingFactor, 
        param->zoomWindowSize * param->oversamplingFactor,
        param->numberWindowDownInChunk, 
        param->numberWindowAcrossInChunk);
    r_corrBatchZoomInOverSampled->allocate();
        
    offsetInit = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetInit->allocate();
		
    offsetZoomIn = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetZoomIn->allocate();
    		
    offsetFinal = new cuArrays<float2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetFinal->allocate();
		
    corrMaxValue = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    corrMaxValue->allocate();
	
    if(param->oversamplingMethod) {
        corrSincOverSampler = new cuSincOverSamplerR2R(param->zoomWindowSize, param->oversamplingFactor, stream);
    }
    else {	
        corrOverSampler= new cuOverSamplerR2R(param->zoomWindowSize, param->zoomWindowSize,
			(param->zoomWindowSize)*param->oversamplingFactor, 
		    (param->zoomWindowSize)*param->oversamplingFactor,
		    param->numberWindowDownInChunk*param->numberWindowAcrossInChunk, 
		    stream);		
	}	     
    if(param->algorithm == 0) {
        cuCorrFreqDomain = new cuFreqCorrelator(
            param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
            param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
            stream);
        cuCorrFreqDomain_OverSampled = new cuFreqCorrelator(
            param->searchWindowSizeHeight, param->searchWindowSizeWidth,
            param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
            stream); 
    }

	

    debugmsg("all objects in chunk are created ...\n");

}
cuAmpcorChunk::~cuAmpcorChunk()
{
    /*
    delete masterChunkRaw;
    delete slaveChunkRaw;
    delete ChunkOffsetDown;
    delete ChunkOffsetAcross;
    delete masterBatchRaw;
    delete slaveBatchRaw;
    delete masterChunkOverSampled;
    delete slaveChunkOverSampled;
    delete masterChunkOverSampler;
    delete slaveChunkOverSampler;
    delete masterChunk;
    delete slaveChunk;
    delete corrChunk;
    delete offsetInit;
    delete zoomInOffset;
    delete offsetFinal;
    delete corrChunkZoomIn;
    delete corrChunkZoomInOverSampled;
    delete corrOverSampler;
    delete corrSincOverSampler;
    delete corrMaxValue;
    if(param->algorithm == 0)
        delete cuCorrFreqDomain;
    */
}
