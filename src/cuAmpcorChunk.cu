#include "hip/hip_runtime.h"
#include "cuAmpcorChunk.h"
#include "cuAmpcorUtil.h"

/**
 * Run ampcor process for a batch of images (a chunk)
 * @param[in] idxDown_  index oIDIVUP(i,j) ((i+j-1)/j)f the chunk along Down/Azimuth direction
 * @param[in] idxAcross_ index of the chunk along Across/Range direction
 */
void cuAmpcorChunk::run(int idxDown_, int idxAcross_)
{
    // set chunk index
    setIndex(idxDown_, idxAcross_);

    // load master image chunk
    loadMasterChunk();

    //std::cout << "load master chunk ok\n";

    cuArraysAbs(c_masterBatchRaw, r_masterBatchRaw, stream);
    cuArraysSubtractMean(r_masterBatchRaw, stream);
    // load slave image chunk
    loadSlaveChunk();
    cuArraysAbs(c_slaveBatchRaw, r_slaveBatchRaw, stream);

    //std::cout << "load slave chunk ok\n";


    //cross correlation for none-oversampled data
    if(param->algorithm == 0) {
        cuCorrFreqDomain->execute(r_masterBatchRaw, r_slaveBatchRaw, r_corrBatchRaw);
    }
    else {
        cuCorrTimeDomain(r_masterBatchRaw, r_slaveBatchRaw, r_corrBatchRaw, stream); //time domain cross correlation
    }
    cuCorrNormalize(r_masterBatchRaw, r_slaveBatchRaw, r_corrBatchRaw, stream);


    // find the maximum location of none-oversampled correlation
    // 41 x 41, if halfsearchrange=20
    //cuArraysMaxloc2D(r_corrBatchRaw, offsetInit, stream);
    cuArraysMaxloc2D(r_corrBatchRaw, offsetInit, r_maxval, stream);

    offsetInit->outputToFile("offsetInit1", stream);

    // Estimation of statistics
    // Author: Minyan Zhong
    // Extraction of correlation surface around the peak
    cuArraysCopyExtractCorr(r_corrBatchRaw, r_corrBatchRawZoomIn, i_corrBatchZoomInValid, offsetInit, stream);

    hipDeviceSynchronize();

    // debug: output the intermediate results
    r_maxval->outputToFile("r_maxval",stream);
    r_corrBatchRaw->outputToFile("r_corrBatchRaw",stream);
    r_corrBatchRawZoomIn->outputToFile("r_corrBatchRawZoomIn",stream);
    i_corrBatchZoomInValid->outputToFile("i_corrBatchZoomInValid",stream);

    // Summation of correlation and data point values
    cuArraysSumCorr(r_corrBatchRawZoomIn, i_corrBatchZoomInValid, r_corrBatchSum, i_corrBatchValidCount, stream);

    // SNR
    cuEstimateSnr(r_corrBatchSum, i_corrBatchValidCount, r_maxval, r_snrValue, stream);

    // Variance
    // cuEstimateVariance(r_corrBatchRaw, offsetInit, r_maxval, r_covValue, stream);

    // Using the approximate estimation to adjust slave image (half search window size becomes only 4 pixels)
    //offsetInit->debuginfo(stream);
    // determine the starting pixel to extract slave images around the max location
    cuDetermineSlaveExtractOffset(offsetInit,
        param->halfSearchRangeDownRaw, // old range
        param->halfSearchRangeAcrossRaw,
        param->halfZoomWindowSizeRaw,  // new range
        param->halfZoomWindowSizeRaw,
        stream);
    //offsetInit->debuginfo(stream);
    // oversample master
    // (deramping now included in oversampler)
    masterBatchOverSampler->execute(c_masterBatchRaw, c_masterBatchOverSampled, param->derampMethod);
    cuArraysAbs(c_masterBatchOverSampled, r_masterBatchOverSampled, stream);
    cuArraysSubtractMean(r_masterBatchOverSampled, stream);

    // extract slave and oversample
    cuArraysCopyExtract(c_slaveBatchRaw, c_slaveBatchZoomIn, offsetInit, stream);
    slaveBatchOverSampler->execute(c_slaveBatchZoomIn, c_slaveBatchOverSampled, param->derampMethod);
    cuArraysAbs(c_slaveBatchOverSampled, r_slaveBatchOverSampled, stream);

    // correlate oversampled images
    if(param->algorithm == 0) {
        cuCorrFreqDomain_OverSampled->execute(r_masterBatchOverSampled, r_slaveBatchOverSampled, r_corrBatchZoomIn);
    }
    else {
        cuCorrTimeDomain(r_masterBatchOverSampled, r_slaveBatchOverSampled, r_corrBatchZoomIn, stream);
    }
    cuCorrNormalize(r_masterBatchOverSampled, r_slaveBatchOverSampled, r_corrBatchZoomIn, stream);

    //std::cout << "debug correlation oversample\n";
    //std::cout << r_masterBatchOverSampled->height << " " << r_masterBatchOverSampled->width << "\n";
    //std::cout << r_slaveBatchOverSampled->height << " " << r_slaveBatchOverSampled->width << "\n";
    //std::cout << r_corrBatchZoomIn->height << " " << r_corrBatchZoomIn->width << "\n";

    // oversample the correlation surface
    cuArraysCopyExtract(r_corrBatchZoomIn, r_corrBatchZoomInAdjust, make_int2(0,0), stream);

    //std::cout << "debug oversampling " << r_corrBatchZoomInAdjust << " " << r_corrBatchZoomInOverSampled << "\n";

    if(param->oversamplingMethod) {
        corrSincOverSampler->execute(r_corrBatchZoomInAdjust, r_corrBatchZoomInOverSampled);
    }
    else {
        corrOverSampler->execute(r_corrBatchZoomInAdjust, r_corrBatchZoomInOverSampled);
    }

    //find the max again

    cuArraysMaxloc2D(r_corrBatchZoomInOverSampled, offsetZoomIn, corrMaxValue, stream);

    // determine the final offset from non-oversampled (pixel) and oversampled (sub-pixel)
    cuSubPixelOffset(offsetInit, offsetZoomIn, offsetFinal,
        param->oversamplingFactor, param->rawDataOversamplingFactor,
        param->halfSearchRangeDownRaw, param->halfSearchRangeAcrossRaw,
        param->halfZoomWindowSizeRaw, param->halfZoomWindowSizeRaw,
        stream);
    //offsetInit->debuginfo(stream);
    //offsetZoomIn->debuginfo(stream);
    //offsetFinal->debuginfo(stream);

    // Do insertion.
    // Offsetfields.
    cuArraysCopyInsert(offsetFinal, offsetImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);

    // Debugging matrix.
    cuArraysCopyInsert(r_corrBatchSum, floatImage1, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
    cuArraysCopyInsert(i_corrBatchValidCount, intImage1, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);

    // Old: save max correlation coefficients.
    //cuArraysCopyInsert(corrMaxValue, snrImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
    // New: save SNR
    cuArraysCopyInsert(r_snrValue, snrImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);

    // Variance.
    cuArraysCopyInsert(r_covValue, covImage, idxDown_*param->numberWindowDownInChunk, idxAcross_*param->numberWindowAcrossInChunk,stream);
}

void cuAmpcorChunk::setIndex(int idxDown_, int idxAcross_)
{
    idxChunkDown = idxDown_;
    idxChunkAcross = idxAcross_;
	idxChunk = idxChunkAcross + idxChunkDown*param->numberChunkAcross;

    if(idxChunkDown == param->numberChunkDown -1) {
		nWindowsDown = param->numberWindowDown - param->numberWindowDownInChunk*(param->numberChunkDown -1);
	}
	else {
		nWindowsDown = param->numberWindowDownInChunk;
	}

	if(idxChunkAcross == param->numberChunkAcross -1) {
		nWindowsAcross = param->numberWindowAcross - param->numberWindowAcrossInChunk*(param->numberChunkAcross -1);
	}
	else {
		nWindowsAcross = param->numberWindowAcrossInChunk;
	}
	//std::cout << "DEBUG setIndex" << idxChunk << " " << nWindowsDown << " " << nWindowsAcross << "\n";

}

/// obtain the starting pixels for each chip
/// @param[in] oStartPixel
///
void cuAmpcorChunk::getRelativeOffset(int *rStartPixel, const int *oStartPixel, int diff)
{
    for(int i=0; i<param->numberWindowDownInChunk; ++i) {
		int iDown = i;
		if(i>=nWindowsDown) iDown = nWindowsDown-1;
        for(int j=0; j<param->numberWindowAcrossInChunk; ++j){
			int iAcross = j;
			if(j>=nWindowsAcross) iAcross = nWindowsAcross-1;
            int idxInChunk = iDown*param->numberWindowAcrossInChunk+iAcross;
            int idxInAll = (iDown+idxChunkDown*param->numberWindowDownInChunk)*param->numberWindowAcross
				+ idxChunkAcross*param->numberWindowAcrossInChunk+iAcross;
            rStartPixel[idxInChunk] = oStartPixel[idxInAll] - diff;
            //fprintf(stderr, "relative offset %d %d %d %d\n", i, j, rStartPixel[idxInChunk], diff);
        }
    }
}

void cuAmpcorChunk::loadMasterChunk()
{

    // we first load the whole chunk of image from cpu to a gpu buffer c(r)_masterChunkRaw
    // then copy to a batch of windows with (nImages, height, width) (leading dimension on the right)

    // get the chunk size to be loaded to gpu
    int startD = param->masterChunkStartPixelDown[idxChunk]; //start pixel down (along height)
    int startA = param->masterChunkStartPixelAcross[idxChunk]; // start pixel across (along width)
    int height =  param->masterChunkHeight[idxChunk]; // number of pixels along height
    int width = param->masterChunkWidth[idxChunk];  // number of pixels along width

    //use cpu to compute the starting positions for each window
    getRelativeOffset(ChunkOffsetDown->hostData, param->masterStartPixelDown, param->masterChunkStartPixelDown[idxChunk]);
    // copy the positions to gpu
    ChunkOffsetDown->copyToDevice(stream);
    // same for the across direction
    getRelativeOffset(ChunkOffsetAcross->hostData, param->masterStartPixelAcross, param->masterChunkStartPixelAcross[idxChunk]);
    ChunkOffsetAcross->copyToDevice(stream);

    // check whether the image is complex (e.g., SLC) or real( e.g. TIFF)
    if(masterImage->isComplex())
    {
        // allocate a gpu buffer to load data from cpu/file
        // try allocate/deallocate the buffer on the fly to save gpu memory 07/09/19
        c_masterChunkRaw = new cuArrays<float2> (param->maxMasterChunkHeight, param->maxMasterChunkWidth);
        c_masterChunkRaw->allocate();

        // load the data from cpu
        masterImage->loadToDevice((void *)c_masterChunkRaw->devData, startD, startA, height, width, stream);
        //std::cout << "debug load master: " << startD << " " <<  startA << " " <<  height << " "  << width << "\n";

        //copy the chunk to a batch format (nImages, height, width)
        // if derampMethod = 0 (no deramp), take amplitudes; otherwise, copy complex data
        if(param->derampMethod == 0) {
            cuArraysCopyToBatchAbsWithOffset(c_masterChunkRaw, param->masterChunkWidth[idxChunk],
                c_masterBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        }
        else {
            cuArraysCopyToBatchWithOffset(c_masterChunkRaw, param->masterChunkWidth[idxChunk],
                c_masterBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        }
        // deallocate the gpu buffer
        delete c_masterChunkRaw;
    }
    // if the image is real
    else {
        r_masterChunkRaw = new cuArrays<float> (param->maxMasterChunkHeight, param->maxMasterChunkWidth);
        r_masterChunkRaw->allocate();

        // load the data from cpu
        masterImage->loadToDevice((void *)r_masterChunkRaw->devData, startD, startA, height, width, stream);

        // copy the chunk (real) to a batch format (complex)
        cuArraysCopyToBatchWithOffsetR2C(r_masterChunkRaw, param->masterChunkWidth[idxChunk],
                c_masterBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        // deallocate the gpu buffer
        delete r_masterChunkRaw;
    }


}

void cuAmpcorChunk::loadSlaveChunk()
{

    //copy to a batch format (nImages, height, width)
    getRelativeOffset(ChunkOffsetDown->hostData, param->slaveStartPixelDown, param->slaveChunkStartPixelDown[idxChunk]);
    ChunkOffsetDown->copyToDevice(stream);
    getRelativeOffset(ChunkOffsetAcross->hostData, param->slaveStartPixelAcross, param->slaveChunkStartPixelAcross[idxChunk]);
    ChunkOffsetAcross->copyToDevice(stream);

    if(slaveImage->isComplex())
    {
        c_slaveChunkRaw = new cuArrays<float2> (param->maxSlaveChunkHeight, param->maxSlaveChunkWidth);
        c_slaveChunkRaw->allocate();

        //load a chunk from mmap to gpu
        slaveImage->loadToDevice(c_slaveChunkRaw->devData,
            param->slaveChunkStartPixelDown[idxChunk],
            param->slaveChunkStartPixelAcross[idxChunk],
            param->slaveChunkHeight[idxChunk],
            param->slaveChunkWidth[idxChunk],
            stream);

        if(param->derampMethod == 0) {
            cuArraysCopyToBatchAbsWithOffset(c_slaveChunkRaw, param->slaveChunkWidth[idxChunk],
                c_slaveBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        }
        else {
           cuArraysCopyToBatchWithOffset(c_slaveChunkRaw, param->slaveChunkWidth[idxChunk],
                c_slaveBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        }
        delete c_slaveChunkRaw;
    }
    else { //real image
        //allocate the gpu buffer
        r_slaveChunkRaw = new cuArrays<float> (param->maxSlaveChunkHeight, param->maxSlaveChunkWidth);
        r_slaveChunkRaw->allocate();

        //load a chunk from mmap to gpu
        slaveImage->loadToDevice(r_slaveChunkRaw->devData,
            param->slaveChunkStartPixelDown[idxChunk],
            param->slaveChunkStartPixelAcross[idxChunk],
            param->slaveChunkHeight[idxChunk],
            param->slaveChunkWidth[idxChunk],
            stream);

        // convert to the batch format
        cuArraysCopyToBatchWithOffsetR2C(r_slaveChunkRaw, param->slaveChunkWidth[idxChunk],
                c_slaveBatchRaw, ChunkOffsetDown->devData, ChunkOffsetAcross->devData, stream);
        delete r_slaveChunkRaw;
    }
}

cuAmpcorChunk::cuAmpcorChunk(cuAmpcorParameter *param_, GDALImage *master_, GDALImage *slave_,
    cuArrays<float2> *offsetImage_, cuArrays<float> *snrImage_, cuArrays<float3> *covImage_, cuArrays<int> *intImage1_, cuArrays<float> *floatImage1_, hipStream_t stream_)

{
    param = param_;
    masterImage = master_;
    slaveImage = slave_;
    offsetImage = offsetImage_;
    snrImage = snrImage_;
    covImage = covImage_;

    intImage1 = intImage1_;
    floatImage1 = floatImage1_;

    stream = stream_;

    // std::cout << "debug Chunk creator " << param->maxMasterChunkHeight << " " << param->maxMasterChunkWidth << "\n";
    // try allocate/deallocate on the fly to save gpu memory 07/09/19
    // c_masterChunkRaw = new cuArrays<float2> (param->maxMasterChunkHeight, param->maxMasterChunkWidth);
    // c_masterChunkRaw->allocate();

    // c_slaveChunkRaw = new cuArrays<float2> (param->maxSlaveChunkHeight, param->maxSlaveChunkWidth);
    // c_slaveChunkRaw->allocate();

    ChunkOffsetDown = new cuArrays<int> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    ChunkOffsetDown->allocate();
    ChunkOffsetDown->allocateHost();
    ChunkOffsetAcross = new cuArrays<int> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    ChunkOffsetAcross->allocate();
    ChunkOffsetAcross->allocateHost();

    c_masterBatchRaw = new cuArrays<float2> (
        param->windowSizeHeightRaw, param->windowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_masterBatchRaw->allocate();

    c_slaveBatchRaw = new cuArrays<float2> (
        param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_slaveBatchRaw->allocate();

    r_masterBatchRaw = new cuArrays<float> (
        param->windowSizeHeightRaw, param->windowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_masterBatchRaw->allocate();

    r_slaveBatchRaw = new cuArrays<float> (
        param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_slaveBatchRaw->allocate();

    c_slaveBatchZoomIn = new cuArrays<float2> (
        param->searchWindowSizeHeightRawZoomIn, param->searchWindowSizeWidthRawZoomIn,
        param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_slaveBatchZoomIn->allocate();

    c_masterBatchOverSampled = new cuArrays<float2> (
			param->windowSizeHeight, param->windowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_masterBatchOverSampled->allocate();

    c_slaveBatchOverSampled = new cuArrays<float2> (
			param->searchWindowSizeHeight, param->searchWindowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    c_slaveBatchOverSampled->allocate();

    r_masterBatchOverSampled = new cuArrays<float> (
			param->windowSizeHeight, param->windowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_masterBatchOverSampled->allocate();

    r_slaveBatchOverSampled = new cuArrays<float> (
			param->searchWindowSizeHeight, param->searchWindowSizeWidth,
			param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    r_slaveBatchOverSampled->allocate();

    masterBatchOverSampler = new cuOverSamplerC2C(
        c_masterBatchRaw->height, c_masterBatchRaw->width, //orignal size
        c_masterBatchOverSampled->height, c_masterBatchOverSampled->width, //oversampled size
        c_masterBatchRaw->count, stream);

    slaveBatchOverSampler = new cuOverSamplerC2C(c_slaveBatchZoomIn->height, c_slaveBatchZoomIn->width,
            c_slaveBatchOverSampled->height, c_slaveBatchOverSampled->width, c_slaveBatchRaw->count, stream);

    r_corrBatchRaw = new cuArrays<float> (
			param->searchWindowSizeHeightRaw-param->windowSizeHeightRaw+1,
			param->searchWindowSizeWidthRaw-param->windowSizeWidthRaw+1,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    r_corrBatchRaw->allocate();

    r_corrBatchZoomIn = new cuArrays<float> (
			param->searchWindowSizeHeight - param->windowSizeHeight+1,
			param->searchWindowSizeWidth - param->windowSizeWidth+1,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    r_corrBatchZoomIn->allocate();

    r_corrBatchZoomInAdjust = new cuArrays<float> (
			param->searchWindowSizeHeight - param->windowSizeHeight,
			param->searchWindowSizeWidth - param->windowSizeWidth,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    r_corrBatchZoomInAdjust->allocate();


    r_corrBatchZoomInOverSampled = new cuArrays<float> (
        param->zoomWindowSize * param->oversamplingFactor,
        param->zoomWindowSize * param->oversamplingFactor,
        param->numberWindowDownInChunk,
        param->numberWindowAcrossInChunk);
    r_corrBatchZoomInOverSampled->allocate();

    offsetInit = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetInit->allocate();

    offsetZoomIn = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetZoomIn->allocate();

    offsetFinal = new cuArrays<float2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    offsetFinal->allocate();

    corrMaxValue = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);
    corrMaxValue->allocate();


    // new arrays due to snr estimation
    std::cout<< "corrRawZoomInHeight: " << param->corrRawZoomInHeight << "\n";
    std::cout<< "corrRawZoomInWidth: " << param->corrRawZoomInWidth << "\n";

    r_corrBatchRawZoomIn = new cuArrays<float> (
			param->corrRawZoomInHeight,
			param->corrRawZoomInWidth,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    r_corrBatchRawZoomIn->allocate();

    i_corrBatchZoomInValid = new cuArrays<int> (
			param->corrRawZoomInHeight,
			param->corrRawZoomInWidth,
			param->numberWindowDownInChunk,
			param->numberWindowAcrossInChunk);
    i_corrBatchZoomInValid->allocate();


    r_corrBatchSum = new cuArrays<float> (
                    param->numberWindowDownInChunk,
                    param->numberWindowAcrossInChunk);
    r_corrBatchSum->allocate();

    i_corrBatchValidCount = new cuArrays<int> (
                        param->numberWindowDownInChunk,
                        param->numberWindowAcrossInChunk);
    i_corrBatchValidCount->allocate();

    i_maxloc = new cuArrays<int2> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    i_maxloc->allocate();

    r_maxval = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_maxval->allocate();

    r_snrValue = new cuArrays<float> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_snrValue->allocate();

    r_covValue = new cuArrays<float3> (param->numberWindowDownInChunk, param->numberWindowAcrossInChunk);

    r_covValue->allocate();

    // end of new arrays

    if(param->oversamplingMethod) {
        corrSincOverSampler = new cuSincOverSamplerR2R(param->zoomWindowSize, param->oversamplingFactor, stream);
    }
    else {
        corrOverSampler= new cuOverSamplerR2R(param->zoomWindowSize, param->zoomWindowSize,
			(param->zoomWindowSize)*param->oversamplingFactor,
		    (param->zoomWindowSize)*param->oversamplingFactor,
		    param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
		    stream);
	}
    if(param->algorithm == 0) {
        cuCorrFreqDomain = new cuFreqCorrelator(
            param->searchWindowSizeHeightRaw, param->searchWindowSizeWidthRaw,
            param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
            stream);
        cuCorrFreqDomain_OverSampled = new cuFreqCorrelator(
            param->searchWindowSizeHeight, param->searchWindowSizeWidth,
            param->numberWindowDownInChunk*param->numberWindowAcrossInChunk,
            stream);
    }



    debugmsg("all objects in chunk are created ...\n");

}
cuAmpcorChunk::~cuAmpcorChunk()
{
    /*
    delete masterChunkRaw;
    delete slaveChunkRaw;
    delete ChunkOffsetDown;
    delete ChunkOffsetAcross;
    delete masterBatchRaw;
    delete slaveBatchRaw;
    delete masterChunkOverSampled;
    delete slaveChunkOverSampled;
    delete masterChunkOverSampler;
    delete slaveChunkOverSampler;
    delete masterChunk;
    delete slaveChunk;
    delete corrChunk;
    delete offsetInit;
    delete zoomInOffset;
    delete offsetFinal;
    delete corrChunkZoomIn;
    delete corrChunkZoomInOverSampled;
    delete corrOverSampler;
    delete corrSincOverSampler;
    delete corrMaxValue;
    if(param->algorithm == 0)
        delete cuCorrFreqDomain;
    */
}
