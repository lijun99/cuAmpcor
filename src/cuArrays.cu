#include "hip/hip_runtime.h"

#include "cuArrays.h"
#include "hipError_t.h"

	template <typename T>
	void cuArrays<T>::allocate()
	{
		checkCudaErrors(hipMalloc((void **)&devData, getByteSize()));
        is_allocated = 1;
	}

    template <typename T>
    void cuArrays<T>::allocateHost()
    {
        hostData = (T *)malloc(getByteSize());
        //checkCudaErrors(hipHostMalloc((void **)&hostData, getByteSize()));
        is_allocatedHost = 1;
    }

	template <typename T>
	void cuArrays<T>::deallocate()
	{
		checkCudaErrors(hipFree(devData));
        is_allocated = 0;
	}

    template <typename T>
	void cuArrays<T>::deallocateHost()
	{
		//checkCudaErrors(hipHostFree(hostData));
        free(hostData);
        is_allocatedHost = 0;
	}

    template <typename T>
	void cuArrays<T>::copyToHost(hipStream_t stream)
	{
        //std::cout << "debug copy " << is_allocatedHost << " " << is_allocated  << " " << getByteSize() << "\n";
		checkCudaErrors(hipMemcpyAsync(hostData, devData, getByteSize(), hipMemcpyDeviceToHost, stream));
	}

    template <typename T>
    void cuArrays<T>::copyToDevice(hipStream_t stream)
	{
		checkCudaErrors(hipMemcpyAsync(devData, hostData, getByteSize(), hipMemcpyHostToDevice, stream));
	}

    template <typename T>
    void cuArrays<T>::setZero(hipStream_t stream)
    {
        checkCudaErrors(hipMemsetAsync(devData, 0, getByteSize(), stream));
    }

	template<>
	void cuArrays<float2>::debuginfo(hipStream_t stream) {
		//std::cout << height << " " << width << " " << count << std::endl;
        //std::cout << height << " " << width << " " << count << std::endl;
        if( !is_allocatedHost)
    		allocateHost();
        copyToHost(stream);

        //hipStreamSynchronize(stream);
        //std::cout << "debug debuginfo " << size << " " << count << " " << stream << "\n";

		int range = std::min(10, size*count);
	
		for(int i=0; i<range; i++)
			std::cout << "(" <<hostData[i].x << ", " << hostData[i].y << ")" ;
		std::cout << std::endl;
        if(size*count>range) {
            for(int i=size*count-range; i<size*count; i++)
                std::cout << "(" <<hostData[i].x << ", " << hostData[i].y << ")" ;
            std::cout << std::endl;
        }
	}


	template<>
	void cuArrays<int2>::debuginfo(hipStream_t stream) {
		//std::cout << height << " " << width << " " << count << std::endl;
        if( !is_allocatedHost)
    		allocateHost();
        copyToHost(stream);
		int range = std::min(10, size*count);
	
		for(int i=0; i<range; i++)
			std::cout << "(" <<hostData[i].x << ", " << hostData[i].y << ")" ;
		std::cout << std::endl;
		if(size*count>range) {
            for(int i=size*count-range; i<size*count; i++)
                std::cout << "(" <<hostData[i].x << ", " << hostData[i].y << ")" ;
            std::cout << std::endl;
        }
	}

	template <>
	void cuArrays<float>::debuginfo(hipStream_t stream) {
		std::cout << height << " " << width << " " << count << std::endl;
        if( !is_allocatedHost)
    		allocateHost();
        copyToHost(stream);
		
		int range = std::min(10, size*count);
	
		for(int i=0; i<range; i++)
			std::cout << "(" <<hostData[i]  << ")" ;
		std::cout << std::endl;
		if(size*count>range) {
            for(int i=size*count-range; i<size*count; i++)
                std::cout << "(" <<hostData[i] << ")" ;
            std::cout << std::endl;
        }
	}

	template<typename T>
	void cuArrays<T>::outputToFile(std::string fn, hipStream_t stream)
	{
        if( !is_allocatedHost)
    		allocateHost();
        copyToHost(stream);
        outputHostToFile(fn);
	}

    template <typename T>
    void cuArrays<T>::outputHostToFile(std::string fn)
	{
		std::ofstream file;
		file.open(fn.c_str(),  std::ios_base::binary);
		file.write((char *)hostData, getByteSize());
		file.close();
	}

	/*
	template<>
	void cuArrays<float>::outputToFile(std::string fn, hipStream_t stream)
	{
		float *data;
		data = (float *)malloc(size*count*sizeof(float));
		hipMemcpyAsync(data, devData, size*count*sizeof(float), hipMemcpyDeviceToHost, stream);
		std::ofstream file;
		file.open(fn.c_str(),  std::ios_base::binary);
		file.write((char *)data, size*count*sizeof(float));
		file.close();
	}*/

	template<>
	void cuArrays<float2>::outputToFile(std::string fn, hipStream_t stream)
	{
		float *data;
		data = (float *)malloc(size*count*sizeof(float2));
		checkCudaErrors(hipMemcpyAsync(data, devData, size*count*sizeof(float2), hipMemcpyDeviceToHost, stream));
		std::ofstream file;
		file.open(fn.c_str(),  std::ios_base::binary);
		file.write((char *)data, size*count*sizeof(float2));
		file.close();
	}

	template<>
	void cuArrays<float3>::outputToFile(std::string fn, hipStream_t stream)
	{
		float *data;
		data = (float *)malloc(size*count*sizeof(float3));
		checkCudaErrors(hipMemcpyAsync(data, devData, size*count*sizeof(float3), hipMemcpyDeviceToHost, stream));
		std::ofstream file;
		file.open(fn.c_str(),  std::ios_base::binary);
		file.write((char *)data, size*count*sizeof(float3));
		file.close();
	}

	template class cuArrays<float>;
	template class cuArrays<float2>;
    template class cuArrays<float3>;
	template class cuArrays<int2>;
    template class cuArrays<int>;
