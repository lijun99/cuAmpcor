#include "hip/hip_runtime.h"
/*
 * cuArraysPadding.cu
 * Padding Utitilies for oversampling
 */

#include "cuAmpcorUtil.h"
#include "float2.h"

//padding zeros in the middle, move quads to corners  
//for raw chunk data oversampling
__global__ void cuArraysPadding_kernel(
	const float2 *image1, const int height1, const int width1,
	float2 *image2, const int height2, const int width2)
{
	int tx = threadIdx.x + blockDim.x*blockIdx.x;
	int ty = threadIdx.y + blockDim.y*blockIdx.y;  
	if(tx < height1/2 && ty < width1/2)
	{
		int tx1 = height1 - 1 - tx;
		int ty1 = width1 -1 -ty;
		int tx2 = height2 -1 -tx;
		int ty2 = width2 -1 -ty;  
		
		//printf("%d %d %d\n", tx, height1, height2); 
		 
		image2[IDX2R(tx, ty, width2)] = image1[IDX2R(tx, ty, width1)];
		image2[IDX2R(tx2, ty, width2)] = image1[IDX2R(tx1, ty, width1)];
		image2[IDX2R(tx, ty2, width2)] = image1[IDX2R(tx, ty1, width1)];
		image2[IDX2R(tx2, ty2, width2)] = image1[IDX2R(tx1, ty1, width1)];
		
	}
}
//tested 
void cuArraysPadding(cuArrays<float2> *image1, cuArrays<float2> *image2, hipStream_t stream)
{
	int ThreadsPerBlock = NTHREADS2D;
	int BlockPerGridx = IDIVUP (image1->height/2, ThreadsPerBlock);
	int BlockPerGridy = IDIVUP (image1->width/2, ThreadsPerBlock);
	dim3 dimBlock(ThreadsPerBlock, ThreadsPerBlock);
	dim3 dimGrid(BlockPerGridx, BlockPerGridy);
	checkCudaErrors(hipMemsetAsync(image2->devData, 0, image2->getByteSize(),stream));
	cuArraysPadding_kernel<<<dimGrid, dimBlock, 0, stream>>>(
		image1->devData, image1->height, image1->width,
		image2->devData, image2->height, image2->width);
	getLastCudaError("cuArraysPadding_kernel");
} 

inline __device__ float2 cmplxMul(float2 c, float a)
{
	return make_float2(c.x*a, c.y*a);
}

//padding for zoomIned correlation oversampling/interpolation 
__global__ void cuArraysPaddingMany_kernel(
	const float2 *image1, const int height1, const int width1, const int size1,
	float2 *image2, const int height2, const int width2, const int size2, const float factor )
{
	int tx = threadIdx.x + blockDim.x*blockIdx.x;
	int ty = threadIdx.y + blockDim.y*blockIdx.y;  
	if(tx < height1/2 && ty < width1/2)
	{  
		
		int tx1 = height1 - 1 - tx;
		int ty1 = width1 -1 -ty;
		int tx2 = height2 -1 -tx;
		int ty2 = width2 -1 -ty;  
		
		int stride1 = blockIdx.z*size1;
		int stride2 = blockIdx.z*size2;
		//printf("%d %d %d\n", tx, height1, height2); 
		
		image2[IDX2R(tx,  ty,  width2)+stride2] = image1[IDX2R(tx,  ty,  width1)+stride1]*factor;
		image2[IDX2R(tx2, ty,  width2)+stride2] = cmplxMul(image1[IDX2R(tx1, ty,  width1)+stride1], factor);
		image2[IDX2R(tx,  ty2, width2)+stride2] = cmplxMul(image1[IDX2R(tx,  ty1, width1)+stride1], factor);
		image2[IDX2R(tx2, ty2, width2)+stride2] = cmplxMul(image1[IDX2R(tx1, ty1, width1)+stride1], factor);
	}
}

void cuArraysPaddingMany(cuArrays<float2> *image1, cuArrays<float2> *image2, hipStream_t stream)
{
	int ThreadsPerBlock = NTHREADS2D;
	int BlockPerGridx = IDIVUP (image1->height/2, ThreadsPerBlock);
	int BlockPerGridy = IDIVUP (image1->width/2, ThreadsPerBlock);
	dim3 dimBlock(ThreadsPerBlock, ThreadsPerBlock, 1);
	dim3 dimGrid(BlockPerGridx, BlockPerGridy, image1->count);
	
	checkCudaErrors(hipMemsetAsync(image2->devData, 0, image2->getByteSize(),stream));
	float factor = 1.0f/image1->size;
	cuArraysPaddingMany_kernel<<<dimGrid, dimBlock, 0, stream>>>(
		image1->devData, image1->height, image1->width, image1->size,
		image2->devData, image2->height, image2->width, image2->size, factor);
	getLastCudaError("cuArraysPadding_kernel");
} 


// convert float to float2(complex)
__global__ void cuArraysR2C_kernel(float *image1, float2 *image2, int size)
{
	int idx =  threadIdx.x + blockDim.x*blockIdx.x;
	if(idx < size)
	{
		image2[idx].x = image1[idx];
		image2[idx].y =  0.0f;
	}
}

//tested
void cuArraysR2C(cuArrays<float> *image1, cuArrays<float2> *image2, hipStream_t stream)
{
	int size = image1->getSize();
	cuArraysR2C_kernel<<<IDIVUP(size, NTHREADS), NTHREADS, 0, stream>>>(image1->devData, image2->devData, size);
	getLastCudaError("cuArraysR2C");
}


// take real part of float2 to float
__global__ void cuArraysC2R_kernel(float2 *image1, float *image2, int size)
{
	int idx =  threadIdx.x + blockDim.x*blockIdx.x;
	if(idx < size)
	{
		image2[idx] = image1[idx].x;
	}
}

//tested
void cuArraysC2R(cuArrays<float2> *image1, cuArrays<float> *image2, hipStream_t stream)
{
	int size = image1->getSize();
	cuArraysC2R_kernel<<<IDIVUP(size, NTHREADS), NTHREADS, 0, stream>>>(image1->devData, image2->devData, size);
	getLastCudaError("cuArraysC2R");
}

// take real part of float2 to float
__global__ void cuArraysAbs_kernel(float2 *image1, float *image2, int size)
{
	int idx =  threadIdx.x + blockDim.x*blockIdx.x;
	if(idx < size)
	{
		image2[idx] = complexAbs(image1[idx]);
	}
}

//tested
void cuArraysAbs(cuArrays<float2> *image1, cuArrays<float> *image2, hipStream_t stream)
{
	int size = image1->getSize();
	cuArraysAbs_kernel<<<IDIVUP(size, NTHREADS), NTHREADS, 0, stream>>>(image1->devData, image2->devData, size);
	getLastCudaError("cuArraysAbs_kernel");
}








