#include "hip/hip_runtime.h"
/*
 * @file  cuArraysPadding.cu
 * @brief Utilities for padding zeros to cuArrays
 */

#include "cuAmpcorUtil.h"
#include "float2.h"

// cuda kernel for cuArraysPadding
__global__ void cuArraysPadding_kernel(
    const float2 *image1, const int height1, const int width1,
    float2 *image2, const int height2, const int width2)
{
    int tx = threadIdx.x + blockDim.x*blockIdx.x;
    int ty = threadIdx.y + blockDim.y*blockIdx.y;
    if(tx < height1/2 && ty < width1/2)
    {
        int tx1 = height1 - 1 - tx;
        int ty1 = width1 -1 -ty;
        int tx2 = height2 -1 -tx;
        int ty2 = width2 -1 -ty;

        image2[IDX2R(tx, ty, width2)] = image1[IDX2R(tx, ty, width1)];
        image2[IDX2R(tx2, ty, width2)] = image1[IDX2R(tx1, ty, width1)];
        image2[IDX2R(tx, ty2, width2)] = image1[IDX2R(tx, ty1, width1)];
        image2[IDX2R(tx2, ty2, width2)] = image1[IDX2R(tx1, ty1, width1)];

    }
}

/**
 * Padding zeros in the middle, move quads to corners
 * @param[in] image1 input images
 * @param[out] image2 output images
 * @note This routine is for a single image, no longer used
 */
void cuArraysPadding(cuArrays<float2> *image1, cuArrays<float2> *image2, hipStream_t stream)
{
    int ThreadsPerBlock = NTHREADS2D;
    int BlockPerGridx = IDIVUP (image1->height/2, ThreadsPerBlock);
    int BlockPerGridy = IDIVUP (image1->width/2, ThreadsPerBlock);
    dim3 dimBlock(ThreadsPerBlock, ThreadsPerBlock);
    dim3 dimGrid(BlockPerGridx, BlockPerGridy);
    // set output image to 0
    checkCudaErrors(hipMemsetAsync(image2->devData, 0, image2->getByteSize(),stream));
    // copy the quads of input images to four corners of the output images
    cuArraysPadding_kernel<<<dimGrid, dimBlock, 0, stream>>>(
        image1->devData, image1->height, image1->width,
        image2->devData, image2->height, image2->width);
    getLastCudaError("cuArraysPadding_kernel");
}

inline __device__ float2 cmplxMul(float2 c, float a)
{
    return make_float2(c.x*a, c.y*a);
}

// cuda kernel for
__global__ void cuArraysPaddingMany_kernel(
    const float2 *image1, const int height1, const int width1, const int size1,
    float2 *image2, const int height2, const int width2, const int size2, const float factor )
{
    int tx = threadIdx.x + blockDim.x*blockIdx.x;
    int ty = threadIdx.y + blockDim.y*blockIdx.y;
    if(tx < height1/2 && ty < width1/2)
    {

        int tx1 = height1 - 1 - tx;
        int ty1 = width1 -1 -ty;
        int tx2 = height2 -1 -tx;
        int ty2 = width2 -1 -ty;

        int stride1 = blockIdx.z*size1;
        int stride2 = blockIdx.z*size2;

        image2[IDX2R(tx,  ty,  width2)+stride2] = image1[IDX2R(tx,  ty,  width1)+stride1]*factor;
        image2[IDX2R(tx2, ty,  width2)+stride2] = cmplxMul(image1[IDX2R(tx1, ty,  width1)+stride1], factor);
        image2[IDX2R(tx,  ty2, width2)+stride2] = cmplxMul(image1[IDX2R(tx,  ty1, width1)+stride1], factor);
        image2[IDX2R(tx2, ty2, width2)+stride2] = cmplxMul(image1[IDX2R(tx1, ty1, width1)+stride1], factor);
    }
}

/**
 * Padding zeros for FFT oversampling
 * @param[in] image1 input images
 * @param[out] image2 output images
 * @note To keep the band center at (0,0), move quads to corners and pad zeros in the middle
 */
void cuArraysPaddingMany(cuArrays<float2> *image1, cuArrays<float2> *image2, hipStream_t stream)
{
    int ThreadsPerBlock = NTHREADS2D;
    int BlockPerGridx = IDIVUP (image1->height/2, ThreadsPerBlock);
    int BlockPerGridy = IDIVUP (image1->width/2, ThreadsPerBlock);
    dim3 dimBlock(ThreadsPerBlock, ThreadsPerBlock, 1);
    dim3 dimGrid(BlockPerGridx, BlockPerGridy, image1->count);

    checkCudaErrors(hipMemsetAsync(image2->devData, 0, image2->getByteSize(),stream));
    float factor = 1.0f/image1->size;
    cuArraysPaddingMany_kernel<<<dimGrid, dimBlock, 0, stream>>>(
        image1->devData, image1->height, image1->width, image1->size,
        image2->devData, image2->height, image2->width, image2->size, factor);
    getLastCudaError("cuArraysPadding_kernel");
}
//end of file








