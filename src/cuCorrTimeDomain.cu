#include "hip/hip_runtime.h"
/* 
 * cuCorrTimetime.cu 
 * correlation between two sets of images in time domain
 */

#include "cuAmpcorUtil.h"

template<const int nthreads, const int NPT>
__global__ void cuArraysCorrTime_kernel(
	const int nImages, 
	const float *templateIn, const int templateNY, const int templateNX, const int templateSize,   
	const float *imageIn, const int imageNY, const int imageNX, const int imageSize,  
    float *resultOut, const int resultNY, const int resultNX, const int resultSize)
{
    __shared__ float shmem[nthreads*(1+NPT)];
    const int tid = threadIdx.x;
    const int bid =  blockIdx.x;
    const int  yc =  blockIdx.y*NPT;
    
    const int       imageIdx = bid;
    const int    imageOffset = imageIdx *    imageSize;
    const int templateOffset = imageIdx * templateSize;
    const int   resultOffset = imageIdx *   resultSize;
    
    const float *   imageD =    imageIn  +    imageOffset + tid;
    const float *templateD = templateIn  + templateOffset + tid;
     float *  resultD =   resultOut +   resultOffset;
    
    const int q  = min(nthreads/resultNX, 4);
    const int nt = nthreads/q;
    const int ty = threadIdx.x / nt;
    const int tx = threadIdx.x - nt * ty;
    
    const int templateNXq = templateNX/q;
    const int jbeg = templateNXq * ty;
    const int jend = ty+1 >= q ? templateNX : templateNXq + jbeg;
    
    float *shTemplate = shmem;
    float *shImage    = shmem + nthreads;
    float *shImage1   = shImage + tx;
    
    float corrCoeff[NPT];
    for (int k = 0; k < NPT; k++)
        corrCoeff[k] = 0.0f;
    
    int iaddr = yc*imageNX;
    

    float img[NPT];
    for (int k = 0; k < NPT-1; k++, iaddr += imageNX)
        img[k] = imageD[iaddr]; 
    for (int taddr = 0; taddr < templateSize; taddr += templateNX, iaddr += imageNX)
    {
        shTemplate[tid] = templateD[taddr];
        img     [NPT-1] =    imageD[iaddr];
        for (int k = 0; k < NPT; k++)
            shImage[tid + nthreads*k] = img[k];
        for (int k = 0; k < NPT-1; k++)
            img[k] = img[k+1];
        __syncthreads();
        
        if (tx < resultNX && ty < q)
        {
#pragma unroll 8  
            for (int j = jbeg; j < jend; j++)
                for (int k = 0; k < NPT; k++)
                    corrCoeff[k] += shTemplate[j]*shImage1[j + nthreads*k];
        }
        __syncthreads();
    }

    for (int k = 0; k < NPT; k++)
        shmem[tid + nthreads*k] = corrCoeff[k];
    __syncthreads();
    
    for (int j = tx + nt; j < nthreads; j += nt)
        for (int k = 0; k < NPT; k++)
            corrCoeff[k] += shmem[j + nthreads*k];
    __syncthreads();
    
    if (tid < resultNX)
    {
        int raddr = yc*resultNX + tid;
        for (int k = 0; k < NPT; k++, raddr += resultNX)
            if (raddr < resultSize)
                resultD[raddr] = corrCoeff[k];
    }
}


void cuCorrTimeDomain(cuArrays<float> *templates,
			   cuArrays<float> *images,
			   cuArrays<float> *results,
			   hipStream_t stream)
{
    /* compute correlation matrix */
    const int nImages = images->count;
    const int imageNX = images->width;
    const int NPT = 8;
    
    
    const dim3 grid(nImages, (results->width-1)/NPT+1, 1);
    //fprintf(stderr, "corrTimeDomain %d %d %d\n", imageNX, templates->height, results->height);
    if      (imageNX <=   64) cuArraysCorrTime_kernel<  64,NPT><<<grid,  64, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <=  128) cuArraysCorrTime_kernel< 128,NPT><<<grid, 128, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <=  192) cuArraysCorrTime_kernel< 192,NPT><<<grid, 192, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <=  256) cuArraysCorrTime_kernel< 256,NPT><<<grid, 256, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <=  384) cuArraysCorrTime_kernel< 384,NPT><<<grid, 384, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <=  512) cuArraysCorrTime_kernel< 512,NPT><<<grid, 512, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <=  640) cuArraysCorrTime_kernel< 640,NPT><<<grid, 640, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <=  768) cuArraysCorrTime_kernel< 768,NPT><<<grid, 768, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <=  896) cuArraysCorrTime_kernel< 896,NPT><<<grid, 896, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else if (imageNX <= 1024) cuArraysCorrTime_kernel<1024,NPT><<<grid,1024, 0, stream>>>(nImages, 
		templates->devData, templates->height, templates->width, templates->size, 
		images->devData, images->height, images->width, images->size,
		results->devData, results->height, results->width, results->size);
    else assert(0);
	getLastCudaError("cuArraysCorrTime error");
}
