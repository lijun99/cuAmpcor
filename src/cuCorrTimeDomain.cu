#include "hip/hip_runtime.h"
/*
 * @file  cuCorrTimetime.cu
 * @brief Correlation between two sets of images in time domain
 *
 * This code is adapted from the nxcor package.
 */

#include "cuAmpcorUtil.h"


// cuda kernel for cuCorrTimeDomain
template<const int nthreads, const int NPT>
__global__ void cuArraysCorrTime_kernel(
    const int nImages,
    const float *templateIn, const int templateNX, const int templateNY, const int templateSize,
    const float *imageIn, const int imageNX, const int imageNY, const int imageSize,
    float *resultOut, const int resultNX, const int resultNY, const int resultSize)
{
    __shared__ float shmem[nthreads*(1+NPT)];
    const int tid = threadIdx.x;
    const int bid =  blockIdx.x;
    const int  yc =  blockIdx.y*NPT;

    const int       imageIdx = bid;
    const int    imageOffset = imageIdx *    imageSize;
    const int templateOffset = imageIdx * templateSize;
    const int   resultOffset = imageIdx *   resultSize;

    const float *   imageD =    imageIn  +    imageOffset + tid;
    const float *templateD = templateIn  + templateOffset + tid;
     float *  resultD =   resultOut +   resultOffset;

    const int q  = min(nthreads/resultNY, 4);
    const int nt = nthreads/q;
    const int ty = threadIdx.x / nt;
    const int tx = threadIdx.x - nt * ty;

    const int templateNYq = templateNY/q;
    const int jbeg = templateNYq * ty;
    const int jend = ty+1 >= q ? templateNY : templateNYq + jbeg;

    float *shTemplate = shmem;
    float *shImage    = shmem + nthreads;
    float *shImage1   = shImage + tx;

    float corrCoeff[NPT];
    for (int k = 0; k < NPT; k++)
        corrCoeff[k] = 0.0f;

    int iaddr = yc*imageNY;


    float img[NPT];
    for (int k = 0; k < NPT-1; k++, iaddr += imageNY)
        img[k] = imageD[iaddr];
    for (int taddr = 0; taddr < templateSize; taddr += templateNY, iaddr += imageNY)
    {
        shTemplate[tid] = templateD[taddr];
        img     [NPT-1] =    imageD[iaddr];
        for (int k = 0; k < NPT; k++)
            shImage[tid + nthreads*k] = img[k];
        for (int k = 0; k < NPT-1; k++)
            img[k] = img[k+1];
        __syncthreads();

        if (tx < resultNY && ty < q)
        {
#pragma unroll 8
            for (int j = jbeg; j < jend; j++)
                for (int k = 0; k < NPT; k++)
                    corrCoeff[k] += shTemplate[j]*shImage1[j + nthreads*k];
        }
        __syncthreads();
    }

    for (int k = 0; k < NPT; k++)
        shmem[tid + nthreads*k] = corrCoeff[k];
    __syncthreads();

    for (int j = tx + nt; j < nthreads; j += nt)
        for (int k = 0; k < NPT; k++)
            corrCoeff[k] += shmem[j + nthreads*k];
    __syncthreads();

    if (tid < resultNY)
    {
        int raddr = yc*resultNY + tid;
        for (int k = 0; k < NPT; k++, raddr += resultNY)
            if (raddr < resultSize)
                resultD[raddr] = corrCoeff[k];
    }
}

/**
 * Perform cross correlation in time domain
 * @param[in] templates Reference images
 * @param[in] images Secondary images
 * @param[out] results Output correlation surface
 * @param[in] stream cudaStream
 */
void cuCorrTimeDomain(cuArrays<float> *templates,
               cuArrays<float> *images,
               cuArrays<float> *results,
               hipStream_t stream)
{
    /* compute correlation matrix */
    const int nImages = images->count;
    const int imageNY = images->width;
    const int NPT = 8;


    const dim3 grid(nImages, (results->width-1)/NPT+1, 1);
    if      (imageNY <=   64) {
        cuArraysCorrTime_kernel<  64,NPT><<<grid,  64, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <=  128)  {
        cuArraysCorrTime_kernel< 128,NPT><<<grid, 128, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <=  192) {
        cuArraysCorrTime_kernel< 192,NPT><<<grid, 192, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <=  256) {
        cuArraysCorrTime_kernel< 256,NPT><<<grid, 256, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <=  384) {
        cuArraysCorrTime_kernel< 384,NPT><<<grid, 384, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
            getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <=  512) {
        cuArraysCorrTime_kernel< 512,NPT><<<grid, 512, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <=  640) {
        cuArraysCorrTime_kernel< 640,NPT><<<grid, 640, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <=  768) {
        cuArraysCorrTime_kernel< 768,NPT><<<grid, 768, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <=  896) {
        cuArraysCorrTime_kernel< 896,NPT><<<grid, 896, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else if (imageNY <= 1024) {
        cuArraysCorrTime_kernel<1024,NPT><<<grid,1024, 0, stream>>>(nImages,
            templates->devData, templates->height, templates->width, templates->size,
            images->devData, images->height, images->width, images->size,
            results->devData, results->height, results->width, results->size);
        getLastCudaError("cuArraysCorrTime error");
    }
    else {
        fprintf(stderr, "The (oversampled) window size along the across direction %d should be smaller than 1024.\n", imageNY);
        throw;
    }
}
// end of file
